#include "cudaUtilities.h"
#include <hipsolver.h>
#include "hipblas.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


const char *cudaGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }

    return "<unknown>";
}



