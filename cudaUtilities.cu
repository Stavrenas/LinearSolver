#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include ""
#include "cudaUtilities.h"
#include "hip/hip_runtime_api.h"

__global__ void floatToDoubleVector(float *left, double *right, int size)
{
    int parts = (float)size / blockDim.x + 1;
    int start = threadIdx.x * parts;
    int end = start + parts;
    if (end > size)
        end = size;
    for (int i = start; i < end; i++)
        right[i] = left[i];
}

// __global__ void matrixVectorMult(int n, int nnz, double *Avalues, int *rowPtr, int *colIdx, double *Vvalues)
// {
//     int total_index = threadIdx.x + blockIdx.x * blockDim.x;

//     int threads = 64;
//     int blocks = 512;

//     int lines_per_block = n / blocks + 1;

//     int startLine = blockIdx.x * lines_per_block;
//     int endLine = (blockIdx.x +1)* lines_per_block;
//     if (endLine > n)
//         endLine = n;

//     int lines_per_thread = lines_per_block / threads + 1;

//     if (total_index % 512 ==  0)
//         printf("total index %d, lines_per_block %d , lines_per_thread %d, blockDim.x %d, start line %d, end line %d\n",total_index, lines_per_block, lines_per_thread, blockDim.x,startLine, endLine);
//     double *results = (double *)malloc(lines_per_thread * sizeof(double));

//     double result = 0.0;
//     // double vector = Vvalues[index];
//     for (int j = 0; j < lines_per_thread; j++)
//     {
//         int line = startLine + threadIdx.x + threads * j;
//         if(line> endLine)
//             line = endLine;
//         int start = rowPtr[line];
//         int end = rowPtr[line + 1];
//         for (int i = start; i < end; i++)
//             result += Avalues[i] * Vvalues[colIdx[i]];

//         results[j] = result;
//     }

//     __syncthreads();

//     for (int j = 0; j < lines_per_thread; j++)
//     {
//         int line = startLine + threadIdx.x + threads * j;
//         Vvalues[line] = results[j];
//     }
// }

__global__ void matrixVectorMult(int n, int nnz, double *Avalues, int *rowPtr, int *colIdx, double *Vvalues, double * result)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < n)
    {
        double dot = 0;
        int row_start = rowPtr[row];
        int row_end = rowPtr[row + 1];
        for (int jj = row_start; jj < row_end; jj++)
            dot += Avalues[jj] * Vvalues[colIdx[jj]];
        result[row] = dot;
    }
}

__global__ void spmv_csr_vector_kernel(const int num_rows, const int *ptr, const int *indices, const double *data, const double *x, double *y)
{
    __shared__ double vals[64];
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x; // global thread index
    int warp_id = thread_id / 32;                          // global warp index
    int lane = thread_id & (32 - 1);                       // thread index within the warp
    // one warp per row
    int row = warp_id;
    if (row < num_rows)
    {
        int row_start = ptr[row];
        int row_end = ptr[row + 1];
        // compute running sum per thread
        vals[threadIdx.x] = 0;
        for (int jj = row_start + lane; jj < row_end; jj += 32)
            vals[threadIdx.x] += data[jj] * x[indices[jj]];
        // parallel reduction in shared memory
        if (lane < 16)
            vals[threadIdx.x] += vals[threadIdx.x + 16];
        if (lane < 8)
            vals[threadIdx.x] += vals[threadIdx.x + 8];
        if (lane < 4)
            vals[threadIdx.x] += vals[threadIdx.x + 4];
        if (lane < 2)
            vals[threadIdx.x] += vals[threadIdx.x + 2];
        if (lane < 1)
            vals[threadIdx.x] += vals[threadIdx.x + 1];
        // first thread writes the result
        if (lane == 0)
            y[row] += vals[threadIdx.x];
    }
}
// __global__ void floatToDoubleVector(float *left, double *right, int size)
// {
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index <size)
//     right[index] = left[index];
// }

void gpuLU(SparseMatrix *mat)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];

    int maxThreads, blocks, threads;
    threads = 256;
    if (nnz > threads)
    {
        maxThreads = threads;
        blocks = nnz / maxThreads + 1;
    }
    else
    {
        blocks = 1;
        maxThreads = nnz;
    }
    blocks = 1;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));

    for (int i = 0; i < nnz; i++)
    {
        host_float_values[i] = mat->values[i];
    }

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    // hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    // hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    float *f_values;
    double *d_values;
    int *rowPtr, *colIdx;

    checkCudaErrors(hipMalloc((void **)&f_values, nnz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(f_values, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // INITIALIZE VARIABLES FOR LU FACTORIZATION
    int pBufferSize;
    void *pBuffer;
    // int structural_zero, numerical_zero;

    hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    csrilu02Info_t LUinfo;
    hipsparseCreateCsrilu02Info(&LUinfo);

    double tole = 0;
    float boost = 1e-8;
    checkCudaErrors(hipsparseScsrilu02_numericBoost(sparseHandle, LUinfo, 1, &tole, &boost));

    // printf("Buffer size..\n");
    // CALCULATE LU FACTORIZATION BUFFER SIZE

    checkCudaErrors(hipsparseScsrilu02_bufferSize(sparseHandle, n, nnz, descrA,
                                                 f_values, rowPtr, colIdx, LUinfo, &pBufferSize));

    checkCudaErrors(hipMalloc(&pBuffer, pBufferSize));
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes
    // printf("Buffer size for LU is %d\n", pBufferSize);

    // printf("Analysis..\n");
    // LU FACTORIZATION ANALYSIS
    checkCudaErrors(hipsparseScsrilu02_analysis(sparseHandle, n, nnz, descrA,
                                               f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    hipsparseStatus_t status;
    int structural_zero;
    status = hipsparseXcsrilu02_zeroPivot(sparseHandle, LUinfo, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

    // printf("Factorization..\n");
    // A = L * U
    checkCudaErrors(hipsparseScsrilu02(sparseHandle, n, nnz, descrA,
                                      f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    // GPU TYPECAST
    floatToDoubleVector<<<blocks, maxThreads>>>(f_values, d_values, nnz);

    hipMemcpy(mat->values, d_values, nnz * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(f_values);
    hipFree(d_values);
    hipFree(rowPtr);
    hipFree(colIdx);

    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyCsrilu02Info(LUinfo);
    hipsparseDestroy(sparseHandle);
}

void sortSparseMatrix(SparseMatrix *mat)
{
    int n = mat->size;
    int nnz = mat->row_idx[n];

    double *values;
    int *rowPtr, *colIdx;
    checkCudaErrors(hipMalloc((void **)&values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));

    checkCudaErrors(hipMemcpy(values, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    double *temp_values;
    int *temp_rowPtr, *temp_colIdx;
    checkCudaErrors(hipMalloc((void **)&temp_values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&temp_rowPtr, (n + 1) * sizeof(int)));

    checkCudaErrors(hipMemcpy(temp_values, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipsparseCreate(&sparseHandle);

    size_t bufferSize;
    void *buffer;
    checkCudaErrors(hipsparseCsr2cscEx2_bufferSize(sparseHandle, n, n, nnz, values, rowPtr, colIdx, temp_values,
                                                  temp_rowPtr, temp_colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                                  HIPSPARSE_CSR2CSC_ALG1, &bufferSize));

    // printf("Buffer size is %d\n", bufferSize);
    checkCudaErrors(hipMalloc(&buffer, bufferSize));

    checkCudaErrors(hipsparseCsr2cscEx2(sparseHandle, n, n, nnz, values, rowPtr, colIdx, temp_values,
                                       temp_rowPtr, temp_colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                       HIPSPARSE_CSR2CSC_ALG1, buffer));

    // RUN TWICE TO GET SORTED MATRIX
    checkCudaErrors(hipsparseCsr2cscEx2(sparseHandle, n, n, nnz, temp_values, temp_rowPtr, temp_colIdx, values,
                                       rowPtr, colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                       HIPSPARSE_CSR2CSC_ALG1, buffer));

    checkCudaErrors(hipMemcpy(mat->values, values, nnz * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mat->row_idx, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mat->col_idx, colIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    hipsparseDestroy(sparseHandle);
    hipFree(values);
    hipFree(rowPtr);
    hipFree(colIdx);
    hipFree(temp_values);
    hipFree(temp_rowPtr);
    hipFree(temp_colIdx);
    hipFree(buffer);
}

void residual(SparseMatrix *mat, Vector *B, double *X)
{
    int n = mat->size;
    int nnz = mat->row_idx[n];

    // INITIALIZE CUSOLVER AND CUBLAS
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    double *Avalues, *rhs, *solution;
    int *rowPtr, *colIdx;
    checkCudaErrors(hipMalloc((void **)&Avalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&solution, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rhs, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));

    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Avalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhs, B->values, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(solution, X, n * sizeof(double), hipMemcpyHostToDevice));

    hipsparseSpMatDescr_t descrA;
    hipsparseCreateCsr(&descrA, n, n, nnz, rowPtr, colIdx, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseDnVecDescr_t descrX, descrB;
    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, solution, HIP_R_64F);

    // INITIALIZE VARIABLES FOR LU SOLVE
    size_t spMvBufferSize;
    void *spMvBuffer;
    double minusOne = -1.0;
    double one = 1.0;
    double bNorm;
    hipblasDnrm2(blasHandle, n, rhs, 1, &bNorm);
    checkCudaErrors(hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrA, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize));
    checkCudaErrors(hipMalloc(&spMvBuffer, spMvBufferSize));
    checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrA, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));

    // RESIDUAL NORM
    double resNorm;
    hipblasDnrm2(blasHandle, n, rhs, 1, &resNorm);
    printf("Residual norm is %e\n", resNorm / bNorm);

    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrB);
    hipFree(Avalues);
    hipFree(rhs);
    hipFree(rowPtr);
    hipFree(colIdx);
    hipFree(solution);
    hipFree(spMvBuffer);
}
