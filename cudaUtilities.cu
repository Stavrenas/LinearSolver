#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include ""
#include "cudaUtilities.h"
#include "hip/hip_runtime_api.h"

__global__ void floatToDoubleVector(float *left, double *right, int size)
{
    int parts = (float)size / blockDim.x + 1;
    int start = threadIdx.x * parts;
    int end = start + parts;
    if (end > size)
        end = size;
    for (int i = start; i < end; i++)
        right[i] = left[i];
}

// __global__ void floatToDoubleVector(float *left, double *right, int size)
// {
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index <size)
//     right[index] = left[index];
// }

void gpuLU(SparseMatrix *mat)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];

    int maxThreads, blocks, threads;
    threads = 256;
    if (nnz > threads)
    {
        maxThreads = threads;
        blocks = nnz / maxThreads + 1;
    }
    else
    {
        blocks = 1;
        maxThreads = nnz;
    }
    blocks = 1;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));

    for (int i = 0; i < nnz; i++)
    {
        host_float_values[i] = mat->values[i];
    }

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    // hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    // hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    float *f_values;
    double *d_values;
    int *rowPtr, *colIdx;

    checkCudaErrors(hipMalloc((void **)&f_values, nnz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(f_values, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // INITIALIZE VARIABLES FOR LU FACTORIZATION
    int pBufferSize;
    void *pBuffer;
    // int structural_zero, numerical_zero;

    hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    csrilu02Info_t LUinfo;
    hipsparseCreateCsrilu02Info(&LUinfo);

    double tole = 0;
    float boost = 1e-8;
    checkCudaErrors(hipsparseScsrilu02_numericBoost(sparseHandle, LUinfo, 1, &tole, &boost));

    // printf("Buffer size..\n");
    // CALCULATE LU FACTORIZATION BUFFER SIZE

    checkCudaErrors(hipsparseScsrilu02_bufferSize(sparseHandle, n, nnz, descrA,
                                                 f_values, rowPtr, colIdx, LUinfo, &pBufferSize));

    checkCudaErrors(hipMalloc(&pBuffer, pBufferSize));
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes
    // printf("Buffer size for LU is %d\n", pBufferSize);

    // printf("Analysis..\n");
    // LU FACTORIZATION ANALYSIS
    checkCudaErrors(hipsparseScsrilu02_analysis(sparseHandle, n, nnz, descrA,
                                               f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    hipsparseStatus_t status;
    int structural_zero;
    status = hipsparseXcsrilu02_zeroPivot(sparseHandle, LUinfo, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

    // printf("Factorization..\n");
    // A = L * U
    checkCudaErrors(hipsparseScsrilu02(sparseHandle, n, nnz, descrA,
                                      f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    // GPU TYPECAST
    floatToDoubleVector<<<blocks, maxThreads>>>(f_values, d_values, nnz);

    hipMemcpy(mat->values, d_values, nnz * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(f_values);
    hipFree(d_values);
    hipFree(rowPtr);
    hipFree(colIdx);

    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyCsrilu02Info(LUinfo);
    hipsparseDestroy(sparseHandle);
}

void sortSparseMatrix(SparseMatrix *mat)
{
    int n = mat->size;
    int nnz = mat->row_idx[n];

    double *values;
    int *rowPtr, *colIdx;
    checkCudaErrors(hipMalloc((void **)&values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));

    checkCudaErrors(hipMemcpy(values, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    double *temp_values;
    int *temp_rowPtr, *temp_colIdx;
    checkCudaErrors(hipMalloc((void **)&temp_values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&temp_rowPtr, (n + 1) * sizeof(int)));

    checkCudaErrors(hipMemcpy(temp_values, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipsparseCreate(&sparseHandle);

    size_t bufferSize;
    void *buffer;
    checkCudaErrors(hipsparseCsr2cscEx2_bufferSize(sparseHandle, n, n, nnz, values, rowPtr, colIdx, temp_values,
                                                  temp_rowPtr, temp_colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                                  HIPSPARSE_CSR2CSC_ALG1, &bufferSize));

    // printf("Buffer size is %d\n", bufferSize);
    checkCudaErrors(hipMalloc(&buffer, bufferSize));

    checkCudaErrors(hipsparseCsr2cscEx2(sparseHandle, n, n, nnz, values, rowPtr, colIdx, temp_values,
                                       temp_rowPtr, temp_colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                       HIPSPARSE_CSR2CSC_ALG1, buffer));

    // RUN TWICE TO GET SORTED MATRIX
    checkCudaErrors(hipsparseCsr2cscEx2(sparseHandle, n, n, nnz, temp_values, temp_rowPtr, temp_colIdx, values,
                                       rowPtr, colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                       HIPSPARSE_CSR2CSC_ALG1, buffer));

    checkCudaErrors(hipMemcpy(mat->values, values, nnz * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mat->row_idx, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mat->col_idx, colIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    hipsparseDestroy(sparseHandle);
    hipFree(values);
    hipFree(rowPtr);
    hipFree(colIdx);
    hipFree(temp_values);
    hipFree(temp_rowPtr);
    hipFree(temp_colIdx);

    hipFree(buffer);
}
