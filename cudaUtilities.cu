#include "hip/hip_runtime.h"
#include "cudaUtilities.h"
#include <hipsolver.h>
#include "hipblas.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void floatToDoubleVector(float *left, double *right)
{
    int position = blockIdx.x * blockDim.x + threadIdx.x;
    right[position] = left[position];
}
