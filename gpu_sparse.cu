#include <cblas.h>
#include <lapacke.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include ""
#include <hip/hip_runtime.h>

extern "C"
{
#include "utilities.h"
#include "read.h"
}

static const char *cudaGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }

    return "<unknown>";
}


int main()
{

    char *matrixName = "Test";
    char *filename = (char *)malloc(40 * sizeof(char));
    char *filenameB = (char *)malloc(40 * sizeof(char));
    char *filenameX = (char *)malloc(40 * sizeof(char));
    char *filenameSol = (char *)malloc(40 * sizeof(char));

    Matrix *mat = (Matrix *)malloc(sizeof(Matrix));

    sprintf(filename, "%s.mtx", matrixName);
    sprintf(filenameB, "%s-B.txt", matrixName);
    sprintf(filenameX, "%s-X.txt", matrixName);
    sprintf(filenameSol, "%s-Solution.txt", matrixName);

    //generateMMMatrix(filename, 100, 1000);
    readMMMatrix(filename, mat);
    // printMatrix(mat);

    double *B = (double *)malloc(mat->size * sizeof(double));
    double *X = (double *)malloc(mat->size * sizeof(double));
    double *Xcalculated = (double *)malloc(mat->size * sizeof(double));

    generateSolutionVector(matrixName, mat);
    readVector(filenameB, mat->size, B);
    readVector(filenameX, mat->size, X);

    // INITIALIZE CUSOLVER
    hipsolverSpHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverSpCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSpSetStream(cusolverHandle, stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tolerance = 1e-38;
    int reorder = 0;
    int *singularity = (int *)malloc(sizeof(int));

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // TRANSFER DATA TO GPU
    double *csrValA, *b, *x;
    int *csrRowPtrA, *csrColIndA;

    hipMalloc((void **)&csrValA, nnz * sizeof(double));
    hipMalloc((void **)&b, n * sizeof(double));
    hipMalloc((void **)&x, n * sizeof(double));
    hipMalloc((void **)&csrRowPtrA, (n + 1) * sizeof(int));
    hipMalloc((void **)&csrColIndA, nnz * sizeof(int));

    hipMemcpy(csrValA, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b, B, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(csrRowPtrA, mat->row_idx, (n + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(csrColIndA, mat->col_idx, nnz * sizeof(double), hipMemcpyHostToDevice);

    // cusolverSpDcsrlsvluHost(
    //     cusolverHandle,
    //     n,
    //     nnz,
    //     descrA,
    //     csrValA,
    //     csrRowPtrA,
    //     csrColIndA,
    //     b,
    //     tolerance,
    //     reorder,
    //     x,
    //     singularity);

    error = cusolverSpDcsrlsvluHost(
        cusolverHandle,
        n,
        nnz,
        descrA,
        mat->values,
        mat->row_idx,
        mat->col_idx,
        B,
        tolerance,
        reorder,
        Xcalculated,
        singularity);

    // hipMemcpy(x, Xcalculated, sizeof(double) * n, hipMemcpyDeviceToHost);

    saveVector(filenameSol, mat->size, Xcalculated);

    hipFree(csrValA);
    hipFree(b);
    hipFree(x);
    hipFree(csrRowPtrA);
    hipFree(csrColIndA);

    hipsolverSpDestroy(cusolverHandle);

    if (checkSolutionThres(n,Xcalculated,X,1e-5))
        printf("Solution is True\n");
    else
        printf("Solution is False\n");

    printf("Singularity is %d\n",singularity);
    printf("Status is %s\n",cudaGetErrorEnum(error));
}