#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include ""
#include "cudaUtilities.h"
#include "hip/hip_runtime_api.h"

extern "C"
{
#include "utilities.h"
#include "read.h"
#include "mklILU.h"
#include "types.h"
}

void gpuLU(SparseMatrix *mat)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];

    int maxThreads, blocks, threads;
    threads = 256;
    if (nnz > threads)
    {
        maxThreads = threads;
        blocks = nnz / maxThreads + 1;
    }
    else
    {
        blocks = 1;
        maxThreads = nnz;
    }
    blocks = 1;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));

    for (int i = 0; i < nnz; i++)
    {
        host_float_values[i] = mat->values[i];
    }

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    // hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    // hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    float *f_values;
    double *d_values;
    int *rowPtr, *colIdx;

    checkCudaErrors(hipMalloc((void **)&f_values, nnz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(f_values, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // INITIALIZE VARIABLES FOR LU FACTORIZATION
    int pBufferSize;
    void *pBuffer;
    // int structural_zero, numerical_zero;

    hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    csrilu02Info_t LUinfo;
    hipsparseCreateCsrilu02Info(&LUinfo);

    double tole = 0;
    float boost = 1e-8;
    checkCudaErrors(hipsparseScsrilu02_numericBoost(sparseHandle, LUinfo, 1, &tole, &boost));

    // printf("Buffer size..\n");
    // CALCULATE LU FACTORIZATION BUFFER SIZE

    checkCudaErrors(hipsparseScsrilu02_bufferSize(sparseHandle, n, nnz, descrA,
                                                 f_values, rowPtr, colIdx, LUinfo, &pBufferSize));

    checkCudaErrors(hipMalloc(&pBuffer, pBufferSize));
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes
    // printf("Buffer size for LU is %d\n", pBufferSize);

    // printf("Analysis..\n");
    // LU FACTORIZATION ANALYSIS
    checkCudaErrors(hipsparseScsrilu02_analysis(sparseHandle, n, nnz, descrA,
                                               f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    hipsparseStatus_t status;
    int structural_zero;
    status = hipsparseXcsrilu02_zeroPivot(sparseHandle, LUinfo, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

    // printf("Factorization..\n");
    // A = L * U
    checkCudaErrors(hipsparseScsrilu02(sparseHandle, n, nnz, descrA,
                                      f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    // GPU TYPECAST
    floatToDoubleVector<<<blocks, maxThreads>>>(f_values, d_values, nnz);

    hipMemcpy(mat->values, d_values, nnz * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(f_values);
    hipFree(d_values);
    hipFree(rowPtr);
    hipFree(colIdx);

    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyCsrilu02Info(LUinfo);
    hipsparseDestroy(sparseHandle);
}

void sortSparseMatrix(SparseMatrix *mat)
{
    int n = mat->size;
    int nnz = mat->row_idx[n];

    double *values;
    int *rowPtr, *colIdx;
    checkCudaErrors(hipMalloc((void **)&values, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));

    checkCudaErrors(hipMemcpy(values, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipsparseCreate(&sparseHandle);

    size_t bufferSize;
    void *buffer;
    checkCudaErrors(hipsparseCsr2cscEx2_bufferSize(sparseHandle, n, n, nnz, values, rowPtr, colIdx, values,
                                                  rowPtr, colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                                  HIPSPARSE_CSR2CSC_ALG1, &bufferSize));

    printf("Buffer size is %d\n", bufferSize);
    checkCudaErrors(hipMalloc(&buffer, bufferSize));

    checkCudaErrors(hipsparseCsr2cscEx2(sparseHandle, n, n, nnz, values, rowPtr, colIdx, values,
                                       rowPtr, colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                       HIPSPARSE_CSR2CSC_ALG1, buffer));

    // RUN TWICE TO GET SORTED MATRIX
    checkCudaErrors(hipsparseCsr2cscEx2(sparseHandle, n, n, nnz, values, rowPtr, colIdx, values,
                                       rowPtr, colIdx, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                       HIPSPARSE_CSR2CSC_ALG1, buffer));

    checkCudaErrors(hipMemcpy(mat->values, values, nnz * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mat->row_idx, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(mat->col_idx, colIdx, nnz * sizeof(int), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();
    hipsparseDestroy(sparseHandle);
    hipFree(values);
    hipFree(rowPtr);
    hipFree(colIdx);
    hipFree(buffer);
}

void solveSystemSparse(SparseMatrix *mat, Vector *B, double *X)
{
    double *Xcalculated = (double *)malloc(mat->size * sizeof(double));

    // INITIALIZE CUSOLVER
    hipsolverSpHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverSpCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSpSetStream(cusolverHandle, stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-8;
    int reorder = 3;
    int singularity;

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    error = cusolverSpDcsrlsvluHost(
        cusolverHandle, n, nnz, descrA,
        mat->values,
        mat->row_idx,
        mat->col_idx,
        B->values,
        tol, // tolerance to determine matrix singularity
        reorder, X, &singularity);

    hipsolverSpDestroy(cusolverHandle);

    printf("Singularity is %d\n", singularity);
    // printf("Status is %s\n",cudaGetErrorEnum(error));
}

void solveSystemSparseIterative(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];
    int maxIters = 5000;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));
    double *zeros = (double *)malloc(n * sizeof(double));

    for (int i = 0; i < n; i++)
    {
        zeros[i] = 0.0;
    }

    for (int i = 0; i < nnz; i++)
    {
        host_float_values[i] = mat->values[i];
    }

    sortSparseMatrix(mat);

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    double *Xcalculated = (double *)malloc(n * sizeof(double));
    double *temp = (double *)malloc(nnz * sizeof(double));
    double *Lvalues, *Uvalues, *Avalues, *solution, *rhs, *rhsCopy, *temp_solutionX, *temp_solutionY;
    float *f_values;
    int *rowPtr, *colIdx, *rowPtrCopy, *colIdxCopy;

    checkCudaErrors(hipMalloc((void **)&Uvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Lvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Avalues, nnz * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rhs, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rhsCopy, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&solution, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionX, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionY, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&f_values, nnz * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdxCopy, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtrCopy, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(colIdxCopy, colIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(f_values, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Avalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));

    // CPU ILU
    printSparseMatrix(mat);
    mklIncompleteLU(mat);
    printSparseMatrix(mat);

    // GPU LU
    // gpuLU(mat);

    // COPY FLOAT B ELEMENTS
    // hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    checkCudaErrors(hipMemcpy(rhs, B->values, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsCopy, rhs, n * sizeof(double), hipMemcpyDeviceToDevice));

    // INIT EMPTY VECTOR
    checkCudaErrors(hipMemcpy(temp_solutionX, zeros, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_solutionY, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToDevice));

    // FREE HOST MEMORY
    free(zeros);

    checkCudaErrors(hipMemcpy(Lvalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Uvalues, Lvalues, nnz * sizeof(double), hipMemcpyDeviceToDevice));

    printf("\ndone converting..\n");

    hipsparseSpMatDescr_t descrL, descrU, descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrL, n, n, nnz, rowPtr, colIdx, Lvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrU, n, n, nnz, rowPtr, colIdx, Uvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    printf("Set attributes..\n");

    hipsparseFillMode_t lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t nonUnit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_FILL_MODE, (void *)&lower, sizeof(lower));
    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&unit, sizeof(unit));

    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, (void *)&upper, sizeof(upper));
    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&nonUnit, sizeof(nonUnit));

    // INITIALIZE B,X,Y VECTOR DESCRIPTORS
    hipsparseDnVecDescr_t descrX, descrY, descrB;

    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_64F);
    hipsparseCreateDnVec(&descrY, n, temp_solutionY, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, temp_solutionX, HIP_R_64F);

    // SETUP TRIANGULAR SOLVER DESCRIPTOR
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseSpSV_createDescr(&spsvDescrL);
    hipsparseSpSV_createDescr(&spsvDescrU);
    double plusOne = 1.0;

    // INITIALIZE VARIABLES FOR LU SOLVE
    size_t spSvBufferSizeL, spSvBufferSizeU;
    void *spSvBufferL, *spSvBufferU;

    printf("SpSv analysisL.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                            descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &spSvBufferSizeL));

    hipMalloc((void **)&spSvBufferL, spSvBufferSizeL);
    printf("spSvBufferSizeL: %ld\n", spSvBufferSizeL);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                          descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, spSvBufferL));

    printf("SpSv analysisU.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                            descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &spSvBufferSizeU));
    hipMalloc((void **)&spSvBufferU, spSvBufferSizeU);
    printf("spSvBufferSizeU: %ld\n", spSvBufferSizeU);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                          descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, spSvBufferU));

    printf("SpSv solve L.. \n");
    // solve L*y = b
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                       descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

    printf("SpSv solve U.. \n");
    // solve U*x = y
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                       descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

    // hipMemcpy(X, temp_solutionY, 3 * sizeof(double), hipMemcpyDeviceToHost);
    // for (int j = 0; j < 3; j++)
    //     printf("%e ", X[j]);
    // printf("\n");

    printf("enter loop\n");

    for (int i = 0; i < maxIters; i++)
    {

        double minusOne = -1.0;
        double one = 1.0;
        size_t spMvBufferSize = 0;
        void *spMvBuffer;

        // CALCULATE RESIDUAL and store it on B vector
        hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize);
        hipMalloc(&spMvBuffer, spMvBufferSize);
        hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer);

        // CUBLAS NORM
        double resNormm, bNorm;
        hipblasDnrm2(blasHandle, n, rhs, 1, &resNormm);
        hipblasDnrm2(blasHandle, n, rhsCopy, 1, &bNorm);

        if ((resNormm / bNorm) < tolerance)
        {
            printf("Iters: %d\n", i);
            break;
        }
        // solve L*y = r : B contains the residual
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                           descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // solve U*c = y
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                           descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));
        // Xn+1 = Xn + Cn
        hipblasDaxpy(blasHandle, n, &one, temp_solutionX, 1, solution, 1);
        hipMemcpy(temp_solutionX, solution, n * sizeof(double), hipMemcpyDeviceToDevice);

        // restore B values
        hipMemcpy(rhs, rhsCopy, n * sizeof(double), hipMemcpyDeviceToDevice);

        if (i % 100 == 0)
        {
            printf("i is %d ", i);
            printf("res Norm is %e, ", resNormm);
            printf("b norm is %f ", bNorm);
            printf("buff is %ld ", spMvBufferSize);
            printf("div is %e \n", resNormm / bNorm);
            // hipMemcpy(X, temp_solutionX, n * sizeof(double), hipMemcpyHostToDevice);
        }
    }

    // TRANSFER SOLUTION TO X VECTOR
    hipMemcpy(X, temp_solutionX, n * sizeof(double), hipMemcpyHostToDevice);

    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrY);
    hipsparseDestroyDnVec(descrB);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseDestroy(sparseHandle);

    hipMemcpy(X, solution, n * sizeof(double), hipMemcpyDeviceToHost);

    // FREE RESOURCES
}

// USED TO READ .mtx FILES AND THE CORRESPONDING rhs.mtx
void solveMtx(int argc, char **argv)
{
    const char *matrixName;
    if (argc == 1)
        matrixName = "data/sherman1";
    else
        matrixName = argv[1];
    char *filename = (char *)malloc(40 * sizeof(char));
    char *filenameB = (char *)malloc(40 * sizeof(char));
    char *filenameSol = (char *)malloc(40 * sizeof(char));

    sprintf(filename, "%s.mtx", matrixName);
    sprintf(filenameB, "%s_rhs1.mtx", matrixName);

    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));

    readSparseMMMatrix(filename, sparse);
    readMMVector(filenameB, B);

    double *X = (double *)malloc(B->size * sizeof(double));

    struct timeval start = tic();

    for (int i = 0; i < 1; i++)
        // solveSystemSparseIterative(sparse, B, X, 1e-12);
        mklIncompleteLU(sparse);

    printf("Gpu time is %f\n", toc(start));

    // saveVector("var/Sparse.txt", B->size, X);
}

// USED TO READ .bin FILES WHICH INCLUDE THE RIGHT HAND SIDE AND THE SOLUTION
void solveBin(int argc, char **argv)
{
    char *matrixName;

    if (argc == 2)
        matrixName = argv[1];
    else
        matrixName = "data/n10k.bin";
    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));
    Vector *Xcorrect = (Vector *)malloc(sizeof(Vector));
    double *X = (double *)malloc(sizeof(double));
    readSystem(matrixName, sparse, B, Xcorrect);

    // printSparseMatrix(sparse);

    struct timeval start = tic();

    for (int i = 0; i < 1; i++)
        solveSystemSparseIterative(sparse, B, X, 1e-12);
    // mklIncompleteLU(sparse);

    printf("Sparse time is %f\n", toc(start));

    saveVector("var/X.txt", B->size, X);
}

int main(int argc, char **argv)
{
    // solveMtx(argc, argv);
    solveBin(argc, argv);
}
