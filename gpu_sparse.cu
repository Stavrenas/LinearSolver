#include "hip/hip_runtime.h"
#include <cblas.h>
#include <lapacke.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include ""
#include <hip/hip_runtime.h>
#include "cudaUtilities.h"
#include "hip/hip_runtime_api.h"

extern "C"
{
#include "utilities.h"
#include "read.h"
}

void solveSystemSparse(SparseMatrix *mat, Vector *B, double *X)
{
    double *Xcalculated = (double *)malloc(mat->size * sizeof(double));

    // INITIALIZE CUSOLVER
    hipsolverSpHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverSpCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSpSetStream(cusolverHandle, stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-8;
    int reorder = 3;
    int singularity;

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    error = cusolverSpDcsrlsvluHost(
        cusolverHandle, n, nnz, descrA,
        mat->values,
        mat->row_idx,
        mat->col_idx,
        B->values,
        tol, // tolerance to determine matrix singularity
        reorder, X, &singularity);

    hipsolverSpDestroy(cusolverHandle);

    printf("Singularity is %d\n", singularity);
    // printf("Status is %s\n",cudaGetErrorEnum(error));
}

void solveSystemSparseIterative(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-8;
    int maxIters = 5;
    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));
    float *host_float_rhs = (float *)malloc(n * sizeof(float));
    double *zeros = (double *)malloc(n * sizeof(double));

    int maxThreads, blocks;
    if (nnz > 512)
    {
        maxThreads = 512;
        blocks = nnz / maxThreads + 1;
    }
    else
    {
        blocks = 1;
        maxThreads = nnz;
    }
    for (int i = 0; i < n; i++)
    {
        host_float_rhs[i] = B->values[i];
        zeros[i] = 0.0;
    }

    for (int i = 0; i < nnz; i++)
    {
        host_float_values[i] = mat->values[i];
    }

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    double *Xcalculated = (double *)malloc(n * sizeof(double));
    float *Xcalculatedf = (float *)malloc(n * sizeof(float));
    double *Lvalues, *Uvalues, *Avalues, *solution, *rhs, *rhsCopy, *temp_solutionX, *temp_solutionY;
    float *f_values, *tempBig, *temp;
    int *rowPtr, *colIdx, *rowPtrCopy, *colIdxCopy;

    hipMalloc((void **)&Lvalues, nnz * sizeof(double));
    hipMalloc((void **)&Uvalues, nnz * sizeof(double));
    hipMalloc((void **)&Avalues, nnz * sizeof(double));
    hipMalloc((void **)&solution, n * sizeof(double));
    hipMalloc((void **)&rhs, n * sizeof(double));
    hipMalloc((void **)&rhsCopy, n * sizeof(double));
    hipMalloc((void **)&temp_solutionX, n * sizeof(double));
    hipMalloc((void **)&temp_solutionY, n * sizeof(double));

    hipMalloc((void **)&temp, n * sizeof(float));
    hipMalloc((void **)&f_values, nnz * sizeof(float));
    hipMalloc((void **)&tempBig, nnz * sizeof(float));

    hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int));
    hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int));
    hipMalloc((void **)&colIdx, nnz * sizeof(int));
    hipMalloc((void **)&colIdxCopy, nnz * sizeof(int));

    // COPY MATRIX A TO DEVICE MEMORY
    hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(rowPtrCopy, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(colIdxCopy, colIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice);

    // COPY FLOAT MATRIX ELEMENTS
    hipMemcpy(f_values, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Avalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice);

    // COPY FLOAT B ELEMENTS
    // hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    hipMemcpy(rhs, B->values, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rhsCopy, rhs, n * sizeof(double), hipMemcpyDeviceToDevice);

    // INIT EMPTY VECTOR
    hipMemcpy(temp_solutionX, zeros, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp_solutionY, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToDevice);

    // FREE HOST MEMORY
    free(host_float_rhs);
    free(zeros);
    // free(host_float_values);

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // INITIALIZE VARIABLES FOR LU FACTORIZATION
    int pBufferSize;
    size_t spSvBufferSizeL, spSvBufferSizeU;
    void *pBuffer, *spSvBufferL, *spSvBufferU;
    // int structural_zero, numerical_zero;

    hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    csrilu02Info_t LUinfo;
    hipsparseCreateCsrilu02Info(&LUinfo);

    double tole = 0;
    float boost = 1e-8;
    hipsparseScsrilu02_numericBoost(sparseHandle, LUinfo, 1, &tole, &boost);

    // CALCULATE LU FACTORIZATION BUFFER SIZE
    checkCudaErrors(hipsparseScsrilu02_bufferSize(sparseHandle, n, nnz, descrA,
                                                 f_values, rowPtr, colIdx, LUinfo, &pBufferSize));

    hipMalloc(&pBuffer, pBufferSize);
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes
    // printf("Buffer size for LU is %d\n",pBufferSize);

    // LU FACTORIZATION ANALYSIS
    checkCudaErrors(hipsparseScsrilu02_analysis(sparseHandle, n, nnz, descrA,
                                               f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    hipsparseStatus_t status;
    int structural_zero;
    status = hipsparseXcsrilu02_zeroPivot(sparseHandle, LUinfo, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

    // A = L * U
    checkCudaErrors(hipsparseScsrilu02(sparseHandle, n, nnz, descrA,
                                      f_values, rowPtr, colIdx, LUinfo, policy, pBuffer));

    // f_values now contain L U matrices

    hipsparseDestroyMatDescr(descrA);
    hipFree(pBuffer);
    hipsparseDestroyCsrilu02Info(LUinfo);

    floatToDoubleVector<<<blocks, maxThreads>>>(f_values, Lvalues);
    hipMemcpy(Uvalues, Lvalues, nnz * sizeof(double), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    hipFree(f_values);

    hipsparseSpMatDescr_t descrL, descrU, descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrL, n, n, nnz, rowPtr, colIdx, Lvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrU, n, n, nnz, rowPtr, colIdx, Uvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseFillMode_t lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t nonUnit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_FILL_MODE, (void *)&lower, sizeof(lower));
    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&unit, sizeof(unit));

    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, (void *)&upper, sizeof(upper));
    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&nonUnit, sizeof(nonUnit));

    // hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, &lower, sizeof(lower)); // THIS WORKS
    // hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, &unit, sizeof(unit));

    // INITIALIZE B,X,Y VECTOR DESCRIPTORS
    hipsparseDnVecDescr_t descrX, descrY, descrB;

    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_64F);
    hipsparseCreateDnVec(&descrY, n, temp_solutionY, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, temp_solutionX, HIP_R_64F);

    // SETUP TRIANGULAR SOLVER DESCRIPTOR
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseSpSV_createDescr(&spsvDescrL);
    hipsparseSpSV_createDescr(&spsvDescrU);
    double plusOne = 1.0;

    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                            descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &spSvBufferSizeL));
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                            descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &spSvBufferSizeU));

    hipMalloc((void **)&spSvBufferU, spSvBufferSizeU);
    hipMalloc((void **)&spSvBufferL, spSvBufferSizeL);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                          descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, spSvBufferL));

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                          descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, spSvBufferU));

    // solve L*y = b
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                       descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

    // solve U*x = y
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                       descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

    // hipMemcpy(X, temp_solutionY, 3 * sizeof(double), hipMemcpyDeviceToHost);
    // for (int j = 0; j < 3; j++)
    //     printf("%e ", X[j]);
    // printf("\n");

    hipMemcpy(X, temp_solutionX, 3 * sizeof(double), hipMemcpyDeviceToHost);
    for (int j = 0; j < 3; j++)
        printf("%e ", X[j]);
    printf("\n");

    // hipMemcpy(X, Uvalues, 5 * sizeof(double), hipMemcpyDeviceToHost);
    // for (int j = 0; j < 5; j++)
    //     printf("%e ", X[j]);
    // printf("\n");

    for (int i = 0; i < maxIters; i++)
    {
        // CALCULATE RESIDUAL and store it on B vector
        double minusOne = -1.0;
        double one = 1.0;
        size_t spMvBufferSize = 0;
        ;
        void *spMvBuffer;

        hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize);
        hipMalloc(&spMvBuffer, spMvBufferSize);
        hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer);

        printf("Res is ");
        hipMemcpy(X, rhs, 3 * sizeof(double), hipMemcpyDeviceToHost);
        for (int j = 0; j < 3; j++)
            printf("%e ", X[j]);
        printf("\n");

        // CUBLAS NORM
        double resNormm, bNorm;
        hipblasDnrm2(blasHandle, n, rhs, 1, &resNormm);
        hipblasDnrm2(blasHandle, n, rhsCopy, 1, &bNorm);
        printf("res Norm is %f, b norm is %f and div is %f , buff is %d  \n", resNormm, bNorm, resNormm / bNorm, spMvBufferSize);

        if ((resNormm / bNorm) < tol)
            break;

        // Xn+1 = Xn + Cn
        hipblasDaxpy(blasHandle, n, &one, temp_solutionX, 1, solution, 1);
        hipMemcpy(temp_solutionX, solution, n * sizeof(double), hipMemcpyDeviceToDevice);

        // solve L*y = r
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                           descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        //  step 7: solve U*c = y
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                           descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

        // restore B values
        hipMemcpy(rhs, rhsCopy, n * sizeof(double), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(X, rhs, n * sizeof(double), hipMemcpyHostToDevice);
    // for (int j = 0; j < 3; j++)
    //     printf("%e ", X[j]);
    // printf("\n");

    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrY);
    hipsparseDestroyDnVec(descrB);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseDestroy(sparseHandle);

    hipMemcpy(X, solution, n * sizeof(double), hipMemcpyDeviceToHost);

    // FREE RESOURCES
}

int main(int argc, char **argv)
{
    char *matrixName;
    if (argc == 1)
        matrixName = "data/sherman1";
    else
        matrixName = argv[1];
    char *filename = (char *)malloc(40 * sizeof(char));
    char *filenameB = (char *)malloc(40 * sizeof(char));
    char *filenameSol = (char *)malloc(40 * sizeof(char));

    sprintf(filename, "%s.mtx", matrixName);
    sprintf(filenameB, "%s_rhs1.mtx", matrixName);

    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));

    readSparseMMMatrix(filename, sparse);
    readMMVector(filenameB, B);

    double *X = (double *)malloc(B->size * sizeof(double));

    struct timeval start = tic();

    // solveSystemSparse(sparse, B, X);
    solveSystemSparseIterative(sparse, B, X, 1e-5);
    printf("Sparse time is %f\n", toc(start));

    readSparseMMMatrix(filename, sparse); // overwrite factorized matrix to get original values for evaluation
    checkSolutionSparse(sparse, B, X, 0); // calculate |Ax-b|
    saveVector("Sparse.txt", B->size, X);
}