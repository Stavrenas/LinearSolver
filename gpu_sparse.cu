#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include ""
#include "cudaUtilities.h"
#include "hip/hip_runtime_api.h"

extern "C"
{
#include "utilities.h"
#include "read.h"
#include "mklILU.h"
#include "types.h"
}


void solveSystemSparse(SparseMatrix *mat, Vector *B, double *X)
{
    double *Xcalculated = (double *)malloc(mat->size * sizeof(double));

    // INITIALIZE CUSOLVER
    hipsolverSpHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverSpCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSpSetStream(cusolverHandle, stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-8;
    int reorder = 3;
    int singularity;

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    error = cusolverSpDcsrlsvluHost(
        cusolverHandle, n, nnz, descrA,
        mat->values,
        mat->row_idx,
        mat->col_idx,
        B->values,
        tol, // tolerance to determine matrix singularity
        reorder, X, &singularity);

    hipsolverSpDestroy(cusolverHandle);

    printf("Singularity is %d\n", singularity);
    // printf("Status is %s\n",cudaGetErrorEnum(error));
}

void solveSystemSparseIterative(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];
    int maxIters = 15000;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));
    double *zeros = (double *)malloc(n * sizeof(double));

    for (int i = 0; i < n; i++)
    {
        zeros[i] = 0.0;
    }

    for (int i = 0; i < nnz; i++)
    {
        host_float_values[i] = mat->values[i];
    }

    sortSparseMatrix(mat);
    
    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    double *Xcalculated = (double *)malloc(n * sizeof(double));
    double *temp = (double *)malloc(nnz * sizeof(double));
    double *Lvalues, *Uvalues, *Avalues, *solution, *rhs, *rhsCopy, *temp_solutionX, *temp_solutionY;
    float *f_values;
    int *rowPtr, *colIdx, *rowPtrCopy, *colIdxCopy;

    checkCudaErrors(hipMalloc((void **)&Uvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Lvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Avalues, nnz * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rhs, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rhsCopy, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&solution, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionX, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionY, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&f_values, nnz * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdxCopy, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtrCopy, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(colIdxCopy, colIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(f_values, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Avalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));

    // CPU ILU

    mklIncompleteLU(mat);


    // GPU LU
    // gpuLU(mat);

    // COPY FLOAT B ELEMENTS
    // hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    checkCudaErrors(hipMemcpy(rhs, B->values, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsCopy, rhs, n * sizeof(double), hipMemcpyDeviceToDevice));

    // INIT EMPTY VECTOR
    checkCudaErrors(hipMemcpy(temp_solutionX, zeros, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_solutionY, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToDevice));

    // FREE HOST MEMORY
    free(zeros);

    checkCudaErrors(hipMemcpy(Lvalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Uvalues, Lvalues, nnz * sizeof(double), hipMemcpyDeviceToDevice));


    hipsparseSpMatDescr_t descrL, descrU, descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrL, n, n, nnz, rowPtr, colIdx, Lvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrU, n, n, nnz, rowPtr, colIdx, Uvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    // printf("Set attributes..\n");

    hipsparseFillMode_t lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t nonUnit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_FILL_MODE, (void *)&lower, sizeof(lower));
    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&unit, sizeof(unit));

    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, (void *)&upper, sizeof(upper));
    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&nonUnit, sizeof(nonUnit));

    // INITIALIZE B,X,Y VECTOR DESCRIPTORS
    hipsparseDnVecDescr_t descrX, descrY, descrB;

    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_64F);
    hipsparseCreateDnVec(&descrY, n, temp_solutionY, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, temp_solutionX, HIP_R_64F);

    // SETUP TRIANGULAR SOLVER DESCRIPTOR
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseSpSV_createDescr(&spsvDescrL);
    hipsparseSpSV_createDescr(&spsvDescrU);
    double plusOne = 1.0;

    // INITIALIZE VARIABLES FOR LU SOLVE
    size_t spSvBufferSizeL, spSvBufferSizeU;
    void *spSvBufferL, *spSvBufferU;

    // printf("SpSv analysisL.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                            descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &spSvBufferSizeL));

    checkCudaErrors(hipMalloc((void **)&spSvBufferL, spSvBufferSizeL));
    // printf("spSvBufferSizeL: %ld\n", spSvBufferSizeL);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                          descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, spSvBufferL));

    // printf("SpSv analysisU.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                            descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &spSvBufferSizeU));
    checkCudaErrors(hipMalloc((void **)&spSvBufferU, spSvBufferSizeU));
    // printf("spSvBufferSizeU: %ld\n", spSvBufferSizeU);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                          descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, spSvBufferU));

    // printf("SpSv solve L.. \n");
    // // solve L*y = b
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                       descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

    // printf("SpSv solve U.. \n");
    // // solve U*x = y
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                       descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

    //printf("enter loop\n");

    for (int i = 0; i < maxIters; i++)
    {

        double minusOne = -1.0;
        double one = 1.0;
        size_t spMvBufferSize = 0;
        void *spMvBuffer;

        // CALCULATE RESIDUAL and store it on B vector
        checkCudaErrors(hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize));
        checkCudaErrors(hipMalloc(&spMvBuffer, spMvBufferSize));
        checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));

        // CUBLAS NORM
        double resNormm, bNorm;
        hipblasDnrm2(blasHandle, n, rhs, 1, &resNormm);
        hipblasDnrm2(blasHandle, n, rhsCopy, 1, &bNorm);

        if ((resNormm / bNorm) < tolerance)
        {
            printf("Iters: %d\n", i);
            break;
        }
        // solve L*y = r : B contains the residual
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                           descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // solve U*c = y
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                           descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));
        // Xn+1 = Xn + Cn
        hipblasDaxpy(blasHandle, n, &one, temp_solutionX, 1, solution, 1);
        checkCudaErrors(hipMemcpy(temp_solutionX, solution, n * sizeof(double), hipMemcpyDeviceToDevice));

        // restore B values
        checkCudaErrors(hipMemcpy(rhs, rhsCopy, n * sizeof(double), hipMemcpyDeviceToDevice));

        if (i % 100 == 0)
        {
            printf("i is %d ", i);
            printf("res Norm is %e, ", resNormm);
            printf("b norm is %e ", bNorm);
            printf("buff is %ld ", spMvBufferSize);
            printf("div is %e \n", resNormm / bNorm);
        }
    }
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // TRANSFER SOLUTION TO X VECTOR
    checkCudaErrors(hipMemcpy(X, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToHost));

    // FREE RESOURCES
    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrY);
    hipsparseDestroyDnVec(descrB);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseDestroy(sparseHandle);




}

// USED TO READ .mtx FILES AND THE CORRESPONDING rhs.mtx
void solveMtx(int argc, char **argv)
{
    const char *matrixName;
    if (argc == 1)
        matrixName = "data/sherman1";
    else
        matrixName = argv[1];
    char *filename = (char *)malloc(40 * sizeof(char));
    char *filenameB = (char *)malloc(40 * sizeof(char));
    char *filenameSol = (char *)malloc(40 * sizeof(char));

    sprintf(filename, "%s.mtx", matrixName);
    sprintf(filenameB, "%s_rhs1.mtx", matrixName);

    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));

    readSparseMMMatrix(filename, sparse);
    readMMVector(filenameB, B);

    double *X = (double *)malloc(B->size * sizeof(double));

    struct timeval start = tic();

    for (int i = 0; i < 1; i++)
        solveSystemSparseIterative(sparse, B, X, 1e-12);

    printf("Gpu time is %f\n", toc(start));

    // saveVector("var/Sparse.txt", B->size, X);
}

// USED TO READ .bin FILES WHICH INCLUDE THE RIGHT HAND SIDE AND THE SOLUTION
void solveBin(int argc, char **argv)
{
    char *matrixName;

    if (argc == 2)
        matrixName = argv[1];
    else
        matrixName = "data/n10k.bin";
    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));
    Vector *Xcorrect = (Vector *)malloc(sizeof(Vector));

    readSystem(matrixName, sparse, B, Xcorrect);

    double *X = (double *)malloc(B->size * sizeof(double));
    // printSparseMatrix(sparse);

    struct timeval start = tic();

    for (int i = 0; i < 1; i++)
        solveSystemSparseIterative(sparse, B, X, 1e-5);

    printf("Sparse time is %f\n", toc(start));

    saveVector("var/X.txt", B->size, X);
}

int main(int argc, char **argv)
{
    //solveMtx(argc, argv);
    solveBin(argc, argv);
}
