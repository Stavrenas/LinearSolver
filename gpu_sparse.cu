#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include ""
#include "cudaUtilities.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"

extern "C"
{
#include "utilities.h"
#include "read.h"
#include "mklILU.h"
#include "types.h"
}

void solveSystemSparseDirect(SparseMatrix *mat, Vector *B, double *X)
{
    double *Xcalculated = (double *)malloc(mat->size * sizeof(double));

    // INITIALIZE CUSOLVER
    hipsolverSpHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverSpCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSpSetStream(cusolverHandle, stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-8;
    int reorder = 3;
    int singularity;

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    error = cusolverSpDcsrlsvluHost(
        cusolverHandle, n, nnz, descrA,
        mat->values,
        mat->row_idx,
        mat->col_idx,
        B->values,
        tol, // tolerance to determine matrix singularity
        reorder, X, &singularity);

    hipsolverSpDestroy(cusolverHandle);

    hipsparseHandle_t sparseHandle = NULL;
    hipsparseCreate(&sparseHandle);

    hipblasHandle_t blasHandle;
    hipblasCreate(&blasHandle);

    double *rowPtrCopy, *colIdxCopy, *d_b, *d_x;
    checkCudaErrors(hipMalloc((void **)&d_b, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_x, n * sizeof(double)));
    checkCudaErrors(hipMemcpy(d_x, X, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, B->values, n * sizeof(double), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMemcpy(rowPtrCopy, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void **)&colIdxCopy, nnz * sizeof(int)));
    checkCudaErrors(hipMemcpy(colIdxCopy, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));

    hipsparseSpMatDescr_t descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, mat->values, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseDnVecDescr_t descrX, descrB;

    hipsparseCreateDnVec(&descrB, n, d_b, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, d_x, HIP_R_64F);

    double minusOne = -1.0;
    double one = 1.0;
    size_t spMvBufferSize = 0;
    void *spMvBuffer;

    checkCudaErrors(hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize));
    checkCudaErrors(hipMalloc(&spMvBuffer, spMvBufferSize));

    // CALCULATE RESIDUAL and store it on B vector
    checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));

    // CUBLAS NORM
    double resNormm;
    hipblasDnrm2(blasHandle, n, d_b, 1, &resNormm);

    printf("Norm  is %e\n", resNormm);
    //  printf("Status is %s\n",cudaGetErrorEnum(error));
}

void solveSystemSparseIterativeSingle(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];
    int maxIters = 15000;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));
    float *zeros = (float *)malloc(n * sizeof(float));

    for (int i = 0; i < n; i++)
        zeros[i] = 0.0;

    sortSparseMatrix(mat);

    for (int i = 0; i < nnz; i++)
        host_float_values[i] = mat->values[i];

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    float *Xcalculated = (float *)malloc(n * sizeof(float));
    float *temp = (float *)malloc(nnz * sizeof(float));
    float *Lvalues, *Uvalues, *Avalues, *solution, *rhs, *rhsCopy, *temp_solutionX, *temp_solutionY;
    int *rowPtr, *colIdx, *rowPtrCopy, *colIdxCopy;

    checkCudaErrors(hipMalloc((void **)&Uvalues, nnz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&Lvalues, nnz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&Avalues, nnz * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&rhs, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&rhsCopy, n * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&solution, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionX, n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionY, n * sizeof(float)));

    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdxCopy, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtrCopy, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(colIdxCopy, colIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(Avalues, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));

    // CPU ILU

    mklIncompleteLU(mat);

    // GPU LU
    // gpuLU(mat);

    // COPY FLOAT B ELEMENTS
    // hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    for (int i = 0; i < n; i++)
        host_float_values[i] = B->values[i];

    checkCudaErrors(hipMemcpy(rhs, host_float_values, n * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsCopy, rhs, n * sizeof(float), hipMemcpyDeviceToDevice));

    // INIT EMPTY VECTOR
    checkCudaErrors(hipMemcpy(temp_solutionX, zeros, n * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_solutionY, temp_solutionX, n * sizeof(float), hipMemcpyDeviceToDevice));

    // FREE HOST MEMORY
    free(zeros);

    for (int i = 0; i < nnz; i++)
        host_float_values[i] = mat->values[i];

    checkCudaErrors(hipMemcpy(Lvalues, host_float_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Uvalues, Lvalues, nnz * sizeof(float), hipMemcpyDeviceToDevice));

    hipsparseSpMatDescr_t descrL, descrU, descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    hipsparseCreateCsr(&descrL, n, n, nnz, rowPtr, colIdx, Lvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    hipsparseCreateCsr(&descrU, n, n, nnz, rowPtr, colIdx, Uvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    // printf("Set attributes..\n");

    hipsparseFillMode_t lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t nonUnit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_FILL_MODE, (void *)&lower, sizeof(lower));
    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&unit, sizeof(unit));

    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, (void *)&upper, sizeof(upper));
    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&nonUnit, sizeof(nonUnit));

    // INITIALIZE B,X,Y VECTOR DESCRIPTORS
    hipsparseDnVecDescr_t descrX, descrY, descrB;

    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_32F);
    hipsparseCreateDnVec(&descrY, n, temp_solutionY, HIP_R_32F);
    hipsparseCreateDnVec(&descrX, n, temp_solutionX, HIP_R_32F);

    // SETUP TRIANGULAR SOLVER DESCRIPTOR
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseSpSV_createDescr(&spsvDescrL);
    hipsparseSpSV_createDescr(&spsvDescrU);
    float plusOne = 1.0;

    // INITIALIZE VARIABLES FOR LU SOLVE
    size_t spSvBufferSizeL, spSvBufferSizeU;
    void *spSvBufferL, *spSvBufferU;

    // printf("SpSv analysisL.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                            descrY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &spSvBufferSizeL));

    checkCudaErrors(hipMalloc((void **)&spSvBufferL, spSvBufferSizeL));
    // printf("spSvBufferSizeL: %ld\n", spSvBufferSizeL);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                          descrY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, spSvBufferL));

    // printf("SpSv analysisU.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                            descrX, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &spSvBufferSizeU));
    checkCudaErrors(hipMalloc((void **)&spSvBufferU, spSvBufferSizeU));
    // printf("spSvBufferSizeU: %ld\n", spSvBufferSizeU);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                          descrX, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, spSvBufferU));

    // printf("SpSv solve L.. \n");
    // // solve L*y = b
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                       descrY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

    // printf("SpSv solve U.. \n");
    // // solve U*x = y
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                       descrX, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

    // printf("enter loop\n");

    float minusOne = -1.0;
    float one = 1.0;
    size_t spMvBufferSize = 0;
    void *spMvBuffer;
    checkCudaErrors(hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize));
    checkCudaErrors(hipMalloc(&spMvBuffer, spMvBufferSize));

    // calculate b norm
    float bNorm;
    hipblasSnrm2(blasHandle, n, rhsCopy, 1, &bNorm);

    struct timeval tempTime;
    float spmvTime, solveTime;
    spmvTime = 0;
    solveTime = 0;

    for (int i = 0; i < maxIters; i++)
    {
        // CALCULATE RESIDUAL and store it on B vector
        tempTime = tic();
        checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));
        hipDeviceSynchronize();
        spmvTime += toc(tempTime);

        // CUBLAS NORM
        float resNormm;
        hipblasSnrm2(blasHandle, n, rhs, 1, &resNormm);

        if ((resNormm / bNorm) < tolerance)
        {
            printf("Iterations: %d\n", i);
            break;
        }

        tempTime = tic();
        // solve L*y = r : B contains the residual
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                           descrY, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // solve U*c = y
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                           descrX, HIP_R_32F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

        hipDeviceSynchronize();
        solveTime += toc(tempTime);

        // Xn+1 = Xn + Cn
        hipblasSaxpy(blasHandle, n, &one, temp_solutionX, 1, solution, 1);
        checkCudaErrors(hipMemcpy(temp_solutionX, solution, n * sizeof(float), hipMemcpyDeviceToDevice));

        // restore B values
        checkCudaErrors(hipMemcpy(rhs, rhsCopy, n * sizeof(float), hipMemcpyDeviceToDevice));

        if (i % 100 == 0)
        {
            printf("i is %d ", i);
            printf("res Norm is %e, ", resNormm);
            printf("b norm is %e ", bNorm);
            printf("buff is %ld ", spMvBufferSize);
            printf("div is %e \n", resNormm / bNorm);
        }
    }
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // TRANSFER SOLUTION TO X VECTOR
    checkCudaErrors(hipMemcpy(host_float_values, temp_solutionX, n * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < n; i++)
        X[i] = host_float_values[i];

    printf("Spmv time is %f and Solve time is %f\n", spmvTime, solveTime);

    // FREE RESOURCES
    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrY);
    hipsparseDestroyDnVec(descrB);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseDestroy(sparseHandle);
}

void solveSystemSparseIterativeDouble(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];
    int maxIters = 15000;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));
    double *zeros = (double *)malloc(n * sizeof(double));

    for (int i = 0; i < n; i++)
        zeros[i] = 0.0;

    sortSparseMatrix(mat);

    for (int i = 0; i < nnz; i++)
        host_float_values[i] = mat->values[i];

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    double *Xcalculated = (double *)malloc(n * sizeof(double));
    double *Lvalues, *Uvalues, *Avalues, *solution, *rhs, *rhsCopy, *temp_solutionX, *temp_solutionY;
    int *rowPtr, *colIdx, *rowPtrCopy, *colIdxCopy;

    checkCudaErrors(hipMalloc((void **)&Uvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Lvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Avalues, nnz * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rhs, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rhsCopy, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&solution, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionX, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionY, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdxCopy, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtrCopy, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(colIdxCopy, colIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(Avalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));

    // CPU ILU

    mklIncompleteLU(mat);

    // GPU LU
    // gpuLU(mat);

    // COPY FLOAT B ELEMENTS
    // hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    checkCudaErrors(hipMemcpy(rhs, B->values, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsCopy, rhs, n * sizeof(double), hipMemcpyDeviceToDevice));

    // INIT EMPTY VECTOR
    checkCudaErrors(hipMemcpy(temp_solutionX, zeros, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_solutionY, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToDevice));

    // FREE HOST MEMORY
    free(zeros);

    checkCudaErrors(hipMemcpy(Lvalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Uvalues, Lvalues, nnz * sizeof(double), hipMemcpyDeviceToDevice));

    hipsparseSpMatDescr_t descrL, descrU, descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrL, n, n, nnz, rowPtr, colIdx, Lvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrU, n, n, nnz, rowPtr, colIdx, Uvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    // printf("Set attributes..\n");

    hipsparseFillMode_t lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t nonUnit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_FILL_MODE, (void *)&lower, sizeof(lower));
    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&unit, sizeof(unit));

    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, (void *)&upper, sizeof(upper));
    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&nonUnit, sizeof(nonUnit));

    // INITIALIZE B,X,Y VECTOR DESCRIPTORS
    hipsparseDnVecDescr_t descrX, descrY, descrB;

    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_64F);
    hipsparseCreateDnVec(&descrY, n, temp_solutionY, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, temp_solutionX, HIP_R_64F);

    // SETUP TRIANGULAR SOLVER DESCRIPTOR
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseSpSV_createDescr(&spsvDescrL);
    hipsparseSpSV_createDescr(&spsvDescrU);
    double plusOne = 1.0;

    // INITIALIZE VARIABLES FOR LU SOLVE
    size_t spSvBufferSizeL, spSvBufferSizeU;
    void *spSvBufferL, *spSvBufferU;

    // printf("SpSv analysisL.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                            descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &spSvBufferSizeL));

    checkCudaErrors(hipMalloc((void **)&spSvBufferL, spSvBufferSizeL));
    // printf("spSvBufferSizeL: %ld\n", spSvBufferSizeL);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                          descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, spSvBufferL));

    // printf("SpSv analysisU.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                            descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &spSvBufferSizeU));
    checkCudaErrors(hipMalloc((void **)&spSvBufferU, spSvBufferSizeU));
    // printf("spSvBufferSizeU: %ld\n", spSvBufferSizeU);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                          descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, spSvBufferU));

    // printf("SpSv solve L.. \n");
    // // solve L*y = b
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                       descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

    // printf("SpSv solve U.. \n");
    // // solve U*x = y
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                       descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

    // printf("enter loop\n");

    double minusOne = -1.0;
    double one = 1.0;
    size_t spMvBufferSize = 0;
    void *spMvBuffer;
    checkCudaErrors(hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize));
    checkCudaErrors(hipMalloc(&spMvBuffer, spMvBufferSize));

    // calculate b norm
    double bNorm;
    hipblasDnrm2(blasHandle, n, rhsCopy, 1, &bNorm);

    struct timeval tempTime;
    float spmvTime, solveTime;
    spmvTime = 0;
    solveTime = 0;

    for (int i = 0; i < maxIters; i++)
    {

        // CALCULATE RESIDUAL and store it on B vector
        tempTime = tic();
        checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));
        hipDeviceSynchronize();
        spmvTime += toc(tempTime);

        // CUBLAS NORM
        double resNormm;
        hipblasDnrm2(blasHandle, n, rhs, 1, &resNormm);

        if ((resNormm / bNorm) < tolerance)
        {
            printf("Iterations: %d\n", i);
            break;
        }

        tempTime = tic();
        // solve L*y = r : B contains the residual
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                           descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // solve U*c = y
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                           descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

        hipDeviceSynchronize();
        solveTime += toc(tempTime);

        // Xn+1 = Xn + Cn
        hipblasDaxpy(blasHandle, n, &one, temp_solutionX, 1, solution, 1);
        checkCudaErrors(hipMemcpy(temp_solutionX, solution, n * sizeof(double), hipMemcpyDeviceToDevice));

        // restore B values
        checkCudaErrors(hipMemcpy(rhs, rhsCopy, n * sizeof(double), hipMemcpyDeviceToDevice));

        if (i % 100 == 0)
        {
            printf("i is %d ", i);
            printf("res Norm is %e, ", resNormm);
            printf("b norm is %e ", bNorm);
            printf("buff is %ld ", spMvBufferSize);
            printf("div is %e \n", resNormm / bNorm);
        }
    }
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // TRANSFER SOLUTION TO X VECTOR
    checkCudaErrors(hipMemcpy(X, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToHost));

    printf("Spmv time is %f and Solve time is %f\n", spmvTime, solveTime);

    // FREE RESOURCES
    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrY);
    hipsparseDestroyDnVec(descrB);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseDestroy(sparseHandle);
}

void solveSystemSparseIterativeGC(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{

    int n = mat->size;
    int nnz = mat->row_idx[n];
    int maxIters = 1500;

    // create float copy of system elements
    float *host_float_values = (float *)malloc(nnz * sizeof(float));
    double *zeros = (double *)malloc(n * sizeof(double));

    for (int i = 0; i < n; i++)
        zeros[i] = 0.0;

    sortSparseMatrix(mat);

    for (int i = 0; i < nnz; i++)
        host_float_values[i] = mat->values[i];

    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle = NULL;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;
    // hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    // ALLOCATE MEMORY
    double *Xcalculated = (double *)malloc(n * sizeof(double));
    double *Lvalues, *Uvalues, *Avalues, *solution, *rhs, *rhsCopy, *temp_solutionX, *temp_solutionY;
    int *rowPtr, *colIdx, *rowPtrCopy, *colIdxCopy;

    checkCudaErrors(hipMalloc((void **)&Uvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Lvalues, nnz * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&Avalues, nnz * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rhs, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&rhsCopy, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&solution, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionX, n * sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&temp_solutionY, n * sizeof(double)));

    checkCudaErrors(hipMalloc((void **)&rowPtr, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&rowPtrCopy, (n + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdx, nnz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&colIdxCopy, nnz * sizeof(int)));

    // COPY MATRIX A TO DEVICE MEMORY
    checkCudaErrors(hipMemcpy(rowPtr, mat->row_idx, (n + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(colIdx, mat->col_idx, nnz * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rowPtrCopy, rowPtr, (n + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(colIdxCopy, colIdx, nnz * sizeof(int), hipMemcpyDeviceToDevice));

    // COPY FLOAT MATRIX ELEMENTS
    checkCudaErrors(hipMemcpy(Avalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));

    // CPU ILU

    mklIncompleteLU(mat);

    // GPU LU
    // gpuLU(mat);

    // COPY FLOAT B ELEMENTS
    // hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    checkCudaErrors(hipMemcpy(rhs, B->values, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(rhsCopy, rhs, n * sizeof(double), hipMemcpyDeviceToDevice));

    // INIT EMPTY VECTOR
    checkCudaErrors(hipMemcpy(temp_solutionX, zeros, n * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(temp_solutionY, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToDevice));

    // FREE HOST MEMORY
    free(zeros);

    checkCudaErrors(hipMemcpy(Lvalues, mat->values, nnz * sizeof(double), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(Uvalues, Lvalues, nnz * sizeof(double), hipMemcpyDeviceToDevice));

    hipsparseSpMatDescr_t descrL, descrU, descrACopy;
    // Create a copy of A to calculate residual r = b - Ax
    hipsparseCreateCsr(&descrACopy, n, n, nnz, rowPtrCopy, colIdxCopy, Avalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrL, n, n, nnz, rowPtr, colIdx, Lvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    hipsparseCreateCsr(&descrU, n, n, nnz, rowPtr, colIdx, Uvalues, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    // printf("Set attributes..\n");

    hipsparseFillMode_t lower = HIPSPARSE_FILL_MODE_LOWER;
    hipsparseDiagType_t unit = HIPSPARSE_DIAG_TYPE_UNIT;
    hipsparseFillMode_t upper = HIPSPARSE_FILL_MODE_UPPER;
    hipsparseDiagType_t nonUnit = HIPSPARSE_DIAG_TYPE_NON_UNIT;

    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_FILL_MODE, (void *)&lower, sizeof(lower));
    hipsparseSpMatSetAttribute(descrL, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&unit, sizeof(unit));

    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_FILL_MODE, (void *)&upper, sizeof(upper));
    hipsparseSpMatSetAttribute(descrU, HIPSPARSE_SPMAT_DIAG_TYPE, (void *)&nonUnit, sizeof(nonUnit));

    // INITIALIZE B,X,Y VECTOR DESCRIPTORS
    hipsparseDnVecDescr_t descrX, descrY, descrB;

    hipsparseCreateDnVec(&descrB, n, rhs, HIP_R_64F);
    hipsparseCreateDnVec(&descrY, n, temp_solutionY, HIP_R_64F);
    hipsparseCreateDnVec(&descrX, n, temp_solutionX, HIP_R_64F);

    // SETUP TRIANGULAR SOLVER DESCRIPTOR
    hipsparseSpSVDescr_t spsvDescrL, spsvDescrU;
    hipsparseSpSV_createDescr(&spsvDescrL);
    hipsparseSpSV_createDescr(&spsvDescrU);
    double plusOne = 1.0;

    // INITIALIZE VARIABLES FOR LU SOLVE
    size_t spSvBufferSizeL, spSvBufferSizeU;
    void *spSvBufferL, *spSvBufferU;

    // printf("SpSv analysisL.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                            descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, &spSvBufferSizeL));

    checkCudaErrors(hipMalloc((void **)&spSvBufferL, spSvBufferSizeL));
    // printf("spSvBufferSizeL: %ld\n", spSvBufferSizeL);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                          descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL, spSvBufferL));

    // printf("SpSv analysisU.. \n");
    checkCudaErrors(hipsparseSpSV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                            descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, &spSvBufferSizeU));
    checkCudaErrors(hipMalloc((void **)&spSvBufferU, spSvBufferSizeU));
    // printf("spSvBufferSizeU: %ld\n", spSvBufferSizeU);

    checkCudaErrors(hipsparseSpSV_analysis(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                          descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU, spSvBufferU));

    // printf("SpSv solve L.. \n");
    // // solve L*y = b
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                       descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

    // printf("SpSv solve U.. \n");
    // // solve U*x = y
    checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                       descrX, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));

    // printf("enter loop\n");

    double minusOne = -1.0;
    double one = 1.0;
    double zero = 0.0;
    size_t spMvBufferSize = 0;
    void *spMvBuffer;
    checkCudaErrors(hipsparseSpMV_bufferSize(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, &spMvBufferSize));
    checkCudaErrors(hipMalloc(&spMvBuffer, spMvBufferSize));

    // calculate b norm
    double bNorm;
    hipblasDnrm2(blasHandle, n, rhsCopy, 1, &bNorm);

    // for conjucate gradient
    double *P, *q, *z;
    hipMalloc((void **)&P, n * sizeof(double));
    hipMalloc((void **)&q, n * sizeof(double));
    hipMalloc((void **)&z, n * sizeof(double));

    hipsparseDnVecDescr_t descrP, descrQ, descrZ;

    hipsparseCreateDnVec(&descrP, n, P, HIP_R_64F);
    hipsparseCreateDnVec(&descrQ, n, q, HIP_R_64F);
    hipsparseCreateDnVec(&descrZ, n, z, HIP_R_64F);

    double Pi = 0.0;
    double Pi_prev = 0.0;
    double beta, alpha;
    double resNormm, qNorm, pNorm, zNorm;

    struct timeval tempTime;
    float spmvTime, solveTime;
    spmvTime = 0;
    solveTime = 0;

    for (int i = 0; i < maxIters; i++)
    {

        // checkCudaErrors(hipblasDswap(blasHandle, n, rhs, 1, rhsCopy, 1)); // swap vectors to calculate residual for convergence (rhs is corelated to descrB)
        // // CALCULATE RESIDUAL and store it on B vector
        // checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &minusOne, descrACopy, descrX, &one, descrB, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));

        // checkCudaErrors(hipblasDswap(blasHandle, n, rhs, 1, rhsCopy, 1)); // restore rhs values (r in GC algorithm)

        // Step 3: solve Az <- r //
        // solve L*y = r : B contains the residual
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrL, descrB,
                                           descrY, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrL));

        // solve U*z = y
        checkCudaErrors(hipsparseSpSV_solve(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &plusOne, descrU, descrY,
                                           descrZ, HIP_R_64F, HIPSPARSE_SPSV_ALG_DEFAULT, spsvDescrU));
        // z is stored on descrz

        // hipblasDnrm2(blasHandle, n, z, 1, &zNorm);
        // printf("z norm is %e ", zNorm);

        // Step 4: pi = r^T * z //

        Pi_prev = Pi;
        checkCudaErrors(hipblasDdot(blasHandle, n, rhs, 1, z, 1, &Pi));
        // printf("Pi is %e ", Pi);

        // Step 5-6: if i == 0 , p <-z //
        if (i == 0)
            hipblasDcopy(blasHandle, n, z, 1, P, 1);

        else
        {
            // Step 7-8: beta <- pi/pi_1 //
            beta = Pi / Pi_prev;
            // printf("beta is %e ", beta);

            // Step 9: z <- z + bp
            hipblasDaxpy(blasHandle, n, &beta, P, 1, z, 1); // result is saved on z
            //  p <-z //
            hipblasDcopy(blasHandle, n, z, 1, P, 1);
        }

        // hipblasDnrm2(blasHandle, n, P, 1, &pNorm);
        // printf("p norm is %e ", pNorm);

        // Step 10-11: compute q <- Ap
        checkCudaErrors(hipsparseSpMV(sparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, descrACopy, descrP, &zero, descrQ, HIP_R_64F, HIPSPARSE_SPMV_CSR_ALG2, spMvBuffer));
        // hipblasDnrm2(blasHandle, n, q, 1, &qNorm);
        // printf("q norm is %e ", qNorm);

        // Step 12: a <- Pi / P^T * q
        double temp;
        checkCudaErrors(hipblasDdot(blasHandle, n, P, 1, q, 1, &temp));
        alpha = Pi / temp;
        // printf("a is %e ", alpha);

        // Step 13: Xn+1 = Xn + a * p
        checkCudaErrors(hipblasDaxpy(blasHandle, n, &alpha, P, 1, temp_solutionX, 1));

        alpha *= -1;
        // Step 14: r <- r - a * q
        hipblasDaxpy(blasHandle, n, &alpha, q, 1, rhs, 1); // result is saved on rhs

        // hipblasDnrm2(blasHandle, n, rhs, 1, &resNormm);
        // printf("res Norm is %e, ", resNormm);

        // RESIDUAL NORM
        hipblasDnrm2(blasHandle, n, rhs, 1, &resNormm);

        if ((resNormm / bNorm) < tolerance)
        {
            printf("Iterations: %d\n", i);
            break;
        }

        if (i % 100 == 0)
        {
            printf("i is %d ", i);
            printf("res Norm is %e, ", resNormm);
            printf("a is %e  beta is %e  ", alpha, beta);
            printf("div is %e \n", resNormm / bNorm);
        }
    }
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // TRANSFER SOLUTION TO X VECTOR
    checkCudaErrors(hipMemcpy(X, temp_solutionX, n * sizeof(double), hipMemcpyDeviceToHost));

    // printf("Spmv time is %f and Solve time is %f\n", spmvTime, solveTime);

    // FREE RESOURCES
    hipsparseDestroyDnVec(descrX);
    hipsparseDestroyDnVec(descrY);
    hipsparseDestroyDnVec(descrB);
    hipsparseSpSV_destroyDescr(spsvDescrL);
    hipsparseSpSV_destroyDescr(spsvDescrU);
    hipsparseDestroy(sparseHandle);
}

int main(int argc, char **argv)
{
    char *matrixName = (char *)malloc(40 * sizeof(char));
    char *temp = (char *)"data/n10k.bin";
    char saveFile[40] = "var/GPUX.txt";

    if (argc == 2)
        strcpy(matrixName, argv[1]);
    else
        strcpy(matrixName, temp);

    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));
    Vector *Xcorrect = (Vector *)malloc(sizeof(Vector));

    if (strstr(matrixName, ".bin"))
        readSystem(matrixName, sparse, B, Xcorrect);

    else if (strstr(matrixName, ".mtx"))
    {
        char *filenameB = (char *)malloc(40 * sizeof(char));
        char name[40];

        strcpy(name, matrixName);
        name[strlen(name) - 4] = (char)'\0';
        sprintf(filenameB, "%s_rhs1.mtx", name);
        readSparseMMMatrix(matrixName, sparse);
        readMMVector(filenameB, B);
    }

    double *X = (double *)malloc(B->size * sizeof(double));

    struct timeval start = tic();

    for (int i = 0; i < 1; i++)
        solveSystemSparseIterativeGC(sparse, B, X, 1e-7);
    // solveSystemSparseDirect(sparse, B, X);

    printf("Sparse time is %f\n", toc(start));

    saveVector(saveFile, B->size, X);
}
