#include "hip/hip_runtime.h"
#include <cblas.h>
#include <lapacke.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hipsolver.h>
#include <hipsolver.h>
#include ""
#include <hip/hip_runtime.h>
#include "cudaUtilities.h"

extern "C"
{
#include "utilities.h"
#include "read.h"
}

void solveSystemSparse(SparseMatrix *mat, Vector *B, double *X)
{
    double *Xcalculated = (double *)malloc(mat->size * sizeof(double));


    // INITIALIZE CUSOLVER
    hipsolverSpHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverSpCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSpSetStream(cusolverHandle, stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-8;
    int reorder = 3;
    int singularity;

    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    error = cusolverSpDcsrlsvluHost(
        cusolverHandle,
        n,
        nnz,
        descrA,
        mat->values,
        mat->row_idx,
        mat->col_idx,
        B->values,
        tol, // tolerance to determine matrix singularity
        reorder,
        X,
        &singularity);

    hipsolverSpDestroy(cusolverHandle);

    printf("Singularity is %d\n", singularity);
    // printf("Status is %s\n",cudaGetErrorEnum(error));
}

void solveSystemSparseIterative(SparseMatrix *mat, Vector *B, double *X, double tolerance)
{
    //create float copy of values
    float *host_float_values = (float *)malloc(mat->size * sizeof(float));
    for(int i =0 ; i< mat->size; i++)
        host_float_values[i] = mat->values[i];
    
    // INITIALIZE CUSOLVER
    hipsparseHandle_t sparseHandle;
    hipblasHandle_t blasHandle;
    hipStream_t stream = NULL;

    hipsparseStatus_t status;

    hipsparseCreate(&sparseHandle);
    hipblasCreate(&blasHandle);
    hipStreamCreate(&stream);

    int n = mat->size;
    int nnz = mat->row_idx[n];
    double tol = 1e-12;
    int reorder = 0;
    int *singularity = (int *)malloc(sizeof(int));

    //ALLOCATE SPACE

    double *Xcalculated = (double *)malloc(n * sizeof(double));
    double *d_values,*rhs, *solution, *temp_solution,*deviceB;
    float *f_values;
    int *rowPtr, *colIdx;

    hipMalloc((void **)&d_values, n * sizeof(double));
    hipMalloc((void **)&rhs, n * sizeof(double));
    hipMalloc((void **)&solution, n * sizeof(double));

    hipMalloc((void **)&temp_solution, n * sizeof(double));
    hipMalloc((void **)&f_values, n * sizeof(float));

    hipMalloc((void **)&rowPtr, n* sizeof(int));
    hipMalloc((void **)&colIdx, n* sizeof(int));
    hipMalloc((void **)&deviceB, n * sizeof(double));

    hipMemcpy(d_values, mat->values, n, hipMemcpyHostToDevice);
    hipMemcpy(f_values, host_float_values, n, hipMemcpyHostToDevice);
    hipMemcpy(rhs, B->values, n, hipMemcpyHostToDevice);
    hipMemcpy(rowPtr, mat->row_idx, n, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B->values, n, hipMemcpyHostToDevice);
    hipMemcpy(colIdx, mat->col_idx, n, hipMemcpyHostToDevice);


    // SETUP MATRIX DESCRIPTOR
    hipsparseMatDescr_t descrA;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    // SETUP LU FACTORIZATION
    csrilu02Info_t LUinfo = 0;
    hipsparseCreateCsrilu02Info(&LUinfo);
    hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

    double norm;
    int max, min;

    hipblasDnrm2(blasHandle, n, deviceB, 1, &norm);
    hipblasIdamax(blasHandle, n, deviceB, 1, &max);
    hipblasIdamin(blasHandle, n, deviceB, 1, &min);
    printf("Norm is %f, max is %f, min is %f\n", norm, B->values[max], B->values[min]);

    hipsparseMatDescr_t descr_L = 0;
    hipsparseMatDescr_t descr_U = 0;
    csrsv2Info_t info_L = 0;
    csrsv2Info_t info_U = 0;
    int pBufferSize_A;
    int pBufferSize_L;
    int pBufferSize_U;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const double alpha = 1.;
    const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_U = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    // step 1: create a descriptor which contains
    // - matrix L is base-0
    // - matrix L is lower triangular
    // - matrix L has unit diagonal
    // - matrix U is base-0
    // - matrix U is upper triangular
    // - matrix U has non-unit diagonal

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for csrilu02 and two info's for csrsv2
    hipsparseCreateCsrilu02Info(&LUinfo);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    // step 3: query how much memory used in csrilu02 and csrsv2, and allocate the buffer
    hipsparseDcsrilu02_bufferSize(sparseHandle, n, nnz,
                                 descrA, d_values, rowPtr, colIdx, LUinfo, &pBufferSize_A);
    hipsparseDcsrsv2_bufferSize(sparseHandle, trans_L, n, nnz,
                               descr_L, d_values, rowPtr, colIdx, info_L, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(sparseHandle, trans_U, n, nnz,
                               descr_U, d_values, rowPtr, colIdx, info_U, &pBufferSize_U);

    pBufferSize = fmax(pBufferSize_A, fmax(pBufferSize_L, pBufferSize_U));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void **)&pBuffer, pBufferSize);

    // step 4: perform analysis of incomplete Cholesky on A
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on U
    // The lower(upper) triangular part of A has the same sparsity pattern as L(U),
    // we can do analysis of csrilu0 and csrsv2 simultaneously.

    hipsparseDcsrilu02_analysis(sparseHandle, n, nnz, descrA,
                               d_values, rowPtr, colIdx, LUinfo,
                               policy, pBuffer);
    status = hipsparseXcsrilu02_zeroPivot(sparseHandle, LUinfo, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
    {
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDcsrsv2_analysis(sparseHandle, trans_L, n, nnz, descr_L,
                             d_values, rowPtr, colIdx,
                             info_L, policy, pBuffer);

    hipsparseDcsrsv2_analysis(sparseHandle, trans_U, n, nnz, descr_U,
                             d_values, rowPtr, colIdx,
                             info_U, policy, pBuffer);

    // step 5: A = L * U
    hipsparseDcsrilu02(sparseHandle, n, nnz, descrA,
                      d_values, rowPtr, colIdx, LUinfo, policy, pBuffer);
    status = hipsparseXcsrilu02_zeroPivot(sparseHandle, LUinfo, &numerical_zero);

    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
    {
        printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    // step 6: solve L*z = x
    hipsparseDcsrsv2_solve(sparseHandle, trans_L, n, nnz, &alpha, descr_L,
                          d_values, rowPtr, colIdx, info_L,
                          rhs, temp_solution, policy, pBuffer);
    //hipsparseSpSV_solve();

    // step 7: solve U*y = z
    hipsparseDcsrsv2_solve(sparseHandle, trans_U, n, nnz, &alpha, descr_U,
                          d_values, rowPtr, colIdx, info_U,
                          temp_solution, solution, policy, pBuffer);

    hipMemcpy(Xcalculated, solution, n, hipMemcpyDeviceToHost);

    // step 6: free resources
    hipFree(pBuffer);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descr_L);
    hipsparseDestroyMatDescr(descr_U);
    hipsparseDestroyCsrilu02Info(LUinfo);
    hipsparseDestroyCsrsv2Info(info_L);
    hipsparseDestroyCsrsv2Info(info_U);
    hipsparseDestroy(sparseHandle);
}

int main(int argc, char **argv)
{
    char *matrixName;
    if (argc == 1)
        matrixName = "data/e20r0000";
    else
        matrixName = argv[1];
    char *filename = (char *)malloc(40 * sizeof(char));
    char *filenameB = (char *)malloc(40 * sizeof(char));
    char *filenameSol = (char *)malloc(40 * sizeof(char));

    sprintf(filename, "%s.mtx", matrixName);
    sprintf(filenameB, "%s_rhs1.mtx", matrixName);

    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));

    readSparseMMMatrix(filename, sparse);
    readMMVector(filenameB, B);

    double *X = (double *)malloc(B->size * sizeof(double));

    struct timeval start = tic();

    // solveSystemSparse(sparse, B, X);
    solveSystemSparseIterative(sparse, B, X, 1e-5);
    printf("Sparse time is %f\n", toc(start));

    readSparseMMMatrix(filename, sparse); // overwrite factorized matrix to get original values for evaluation
    checkSolutionSparse(sparse, B, X, 0); // calculate |Ax-b|
    saveVector("Sparse.txt", B->size, X);
}