#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "hipblas.h"
#include ""

extern "C"{
#include "utilities.h"
}


static const char *cudaGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }

    return "<unknown>";
}

void Cholesky(double **A, double *B, double *X, int size){

    double * Aserialized = (double *)malloc(size * size * sizeof(double));
    serializeMatrix(size, A, Aserialized);

    // INITIALIZE CUSOLVER
    hipsolverHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    hipsolverDnCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSetStream(cusolverHandle, stream);

    int info;
    int *d_info = nullptr; /* device error info */

    int Lwork = 0;               /* size of workspace */
    double *Workspace = nullptr; /* device workspace */

    /* step 2: COPY MATRICES TO DEVICE */
    double *Bcuda, *Acuda, *Xcuda;

    hipMalloc((void **)&Acuda, size * size * sizeof(double));
    hipMalloc((void **)&Bcuda, size * sizeof(double));
    hipMalloc((void **)&Xcuda, size * sizeof(double));

    hipMemcpy(Acuda, Aserialized, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Bcuda, B, size * sizeof(double), hipMemcpyHostToDevice);

    /* step 3: query working space */
    hipsolverDnDpotrf_bufferSize(cusolverHandle, uplo, size, Acuda, size, &Lwork);
    hipMalloc(&Workspace, sizeof(double) * Lwork);
    hipMalloc((void **)&d_info, sizeof(int));

    /* step 4: Cholesky factorization */
    hipsolverDnDpotrf(cusolverHandle, uplo, size, Acuda, size, Workspace, Lwork, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    if (0 > info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    else if (info > 0)
        exit(1);

    /* step 5: solve A*X = b */
    hipsolverDnDpotrs(cusolverHandle, uplo, size, 1, Acuda, size, Bcuda, size, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    if (0 > info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    hipMemcpy(X, Bcuda, sizeof(double) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* free resources */
    hipFree(Acuda);
    hipFree(Bcuda);
    hipFree(Xcuda);
    hipFree(d_info);
    hipFree(Workspace);

    // hipblasDestroy(handle);
    hipsolverDnDestroy(cusolverHandle);
    hipDeviceReset();

    free(A);
    free(B);
    free(X);
    free(Aserialized);
}

void iterativeRefinement(double **A, double *B, double *X, int size){

    double * Aserialized = (double *)malloc(size * size * sizeof(double));
    serializeMatrix(size, A, Aserialized);

    // INITIALIZE CUSOLVER
    hipsolverHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverDnCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSetStream(cusolverHandle, stream);

    int info, *dipiv;
    int *d_info = nullptr; /* device error info */
    int *iters = (int*)malloc(sizeof(int));

    size_t Lwork = 0;            /* size of workspace */
    void *Workspace = nullptr; /* device workspace */

    /* step 2: COPY MATRICES TO DEVICE */
    double *Bcuda, *Acuda, *Xcuda;

    hipMalloc((void **)&Acuda, size * size * sizeof(double));
    hipMalloc((void **)&Bcuda, size * sizeof(double));
    hipMalloc((void **)&Xcuda, size * sizeof(double));

    hipMalloc((void **)&d_info, sizeof(int));
    hipMalloc((void **)&dipiv, size * size *sizeof(int));

    hipMemcpy(Acuda, Aserialized, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Bcuda, B, size * sizeof(double), hipMemcpyHostToDevice);

    /* step 3: query working space */
    error = cusolverDnDHgesv_bufferSize(cusolverHandle, size, 1, Acuda, size, dipiv, Bcuda, size, Xcuda, size, Workspace, &Lwork);
    hipMalloc(&Workspace, sizeof(double) * Lwork);
    //printf("Error: %s\n", cudaGetErrorEnum(error));

    /* step 4: Cholesky factorization */
    error = cusolverDnDHgesv(cusolverHandle, size, 1, Acuda, size, dipiv, Bcuda, size, Xcuda, size, Workspace, Lwork, iters, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    //printf("Error: %s\n", cudaGetErrorEnum(error));

    if (0 != info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    hipMemcpy(X, Xcuda, sizeof(double) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* free resources */
    hipFree(Acuda);
    hipFree(Bcuda);
    hipFree(Xcuda);
    hipFree(d_info);
    hipFree(Workspace);

    // hipblasDestroy(handle);
    hipsolverDnDestroy(cusolverHandle);
    hipDeviceReset();

    free(Aserialized);

}

int main(){

    int size = 273;
    double **A, *B, *Xcalculated, *X;

    // ALLOCATE MEMORY
    A = (double **)malloc(size * sizeof(double *));
    B = (double *)malloc(size * sizeof(double));
    Xcalculated = (double *)malloc(size * sizeof(double));
    X = (double *)malloc(size * sizeof(double));

    for (int i = 0; i < size; i++)
        A[i] = (double *)malloc(size * sizeof(double));

    // READ MATRICES
    readSquareMatrix("A.txt", size, A);
    readVector("B.txt", size, B);
    readVector("X.txt", size, X);

    struct timeval start = tic();

    iterativeRefinement(A,B,Xcalculated,size);

    printf("Iterative refinement time is %f\n",toc(start));

    if (checkSolution(size, X, Xcalculated))
        printf("Solution is True\n");
    else
        printf("Solution is False\n");

    start = tic();

    Cholesky(A,B,Xcalculated,size);

    printf("Cholesky factorization time is %f\n",toc(start));

    if (checkSolution(size, X, Xcalculated) == 1)
        printf("Solution is True\n");
    else
        printf("Solution is False\n");


}