#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "hipblas.h"
#include ""

extern "C"
{
#include "utilities.h"
#include "read.h"
}

static const char *cudaGetErrorEnum(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "HIPSOLVER_STATUS_SUCCESS";
    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";
    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";
    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";
    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";
    case HIPSOLVER_STATUS_MAPPING_ERROR:
        return "HIPSOLVER_STATUS_MAPPING_ERROR";
    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";
    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";
    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSOLVER_STATUS_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_NOT_SUPPORTED ";
    case HIPSOLVER_STATUS_ZERO_PIVOT:
        return "HIPSOLVER_STATUS_ZERO_PIVOT";
    case CUSOLVER_STATUS_INVALID_LICENSE:
        return "CUSOLVER_STATUS_INVALID_LICENSE";
    }

    return "<unknown>";
}

void iterativeRefinementGeneral(DenseMatrix *A, Vector *B, double *X)
{

    // INITIALIZE CUSOLVER
    hipsolverHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;
    cusolverDnIRSParams_t params;
    cusolverDnIRSInfos_t infos;

    hipsolverDnCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSetStream(cusolverHandle, stream);
    cusolverDnIRSInfosCreate(&infos);

    /* SET ITERATIVE REFINEMENT PARAMETERS*/
    cusolverDnIRSParamsCreate(&params);
    cusolverDnIRSParamsSetSolverPrecisions(params, CUSOLVER_R_64F, CUSOLVER_R_32F); // main and lowest solver precision
    cusolverDnIRSParamsSetRefinementSolver(params, CUSOLVER_IRS_REFINE_CLASSICAL);
    cusolverDnIRSParamsSetTol(params, 1e-8);
    /*This function sets the tolerance for the refinement solver. By default it is such that all the RHS satisfy:

        RNRM < SQRT(N)*XNRM*ANRM*EPS*BWDMAX where
    RNRM is the infinity-norm of the residual
    XNRM is the infinity-norm of the solution
    ANRM is the infinity-operator-norm of the matrix A
    EPS is the machine epsilon for the Inputs/Outputs datatype that matches LAPACK <X>LAMCH('Epsilon')
    BWDMAX, the value BWDMAX is fixed to 1.0
    */


    cusolverDnIRSParamsSetTolInner(params, 1e-10);    // default value is 1e-4
    cusolverDnIRSParamsSetMaxIters(params, 50);      // default value is 50
    cusolverDnIRSParamsSetMaxItersInner(params, 50); // default value is 50
    cusolverDnIRSParamsDisableFallback(params); //by default enabled

    /* SET ITERATIVE REFINEMENT INFOS*/
    int max_iters, n_iters, outer_iters;
    cusolverDnIRSInfosGetMaxIters(infos, &max_iters);

    int info, sides, size;
    int *d_info = nullptr; /* device error info */
    int *iters = (int *)malloc(sizeof(int));

    size_t Lwork = 0;          /* size of workspace */
    void *Workspace = nullptr; /* device workspace */

    /* step 2: COPY MATRICES TO DEVICE */
    double *Bcuda, *Acuda, *Xcuda;
    size = A->size;

    hipMalloc((void **)&Acuda, size * size * sizeof(double));
    hipMalloc((void **)&Bcuda, size * sizeof(double));
    hipMalloc((void **)&Xcuda, size * sizeof(double));

    hipMalloc((void **)&d_info, sizeof(int));

    hipMemcpy(Acuda, A->values, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Bcuda, B->values, size * sizeof(double), hipMemcpyHostToDevice);

    int n = size;
    int nrhs = 1;
    sides = 1;
    /* step 3: query working space */
    error = cusolverDnIRSXgesv_bufferSize(cusolverHandle, params, n, nrhs, &Lwork);
    hipMalloc(&Workspace, Lwork);
    // printf("Error: %s\n", cudaGetErrorEnum(error));

    /* step 4: Iterative Refinement solution */
    error = cusolverDnIRSXgesv(cusolverHandle, params, infos, size, sides, Acuda, size, Bcuda, size, Xcuda, size, Workspace, Lwork, iters, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    // printf("Error: %s\n", cudaGetErrorEnum(error));

    if (0 != info)
    {
        if (info < 0)
            printf("%d-th parameter is wrong \n", -info);
        else
            printf("U(%d,%d) is exactly zero \n", info, info);
        exit(1);
    }

    //cusolverDnIRSInfosGetOuterNiters(infos, &outer_iters);
    cusolverDnIRSInfosGetNiters(infos, &n_iters);

    printf("Iters: %d\n", n_iters);
    printf("Size of Workspace: %d\n", Lwork);

    hipMemcpy(X, Xcuda, sizeof(double) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* free resources */
    hipFree(Acuda);
    hipFree(Bcuda);
    hipFree(Xcuda);
    hipFree(d_info);
    hipFree(Workspace);

    // hipblasDestroy(handle);
    hipsolverDnDestroy(cusolverHandle);
    hipDeviceReset();
}

int main(int argc, char *argv[])
{

    char *matrixName;
    if (argc == 1)
        matrixName = "data/e20r0000";
    else
        matrixName = argv[1];
    char *filename = (char *)malloc(40 * sizeof(char));
    char *filenameB = (char *)malloc(40 * sizeof(char));
    char *filenameSol = (char *)malloc(40 * sizeof(char));

    sprintf(filename, "%s.mtx", matrixName);
    sprintf(filenameB, "%s_rhs1.mtx", matrixName);

    SparseMatrix *sparse = (SparseMatrix *)malloc(sizeof(SparseMatrix));
    DenseMatrix *dense = (DenseMatrix *)malloc(sizeof(DenseMatrix));
    Vector *B = (Vector *)malloc(sizeof(Vector));

    readSparseMMMatrix(filename, sparse);
    readMMVector(filenameB, B);
    sparseToDense(sparse, dense);

    double *X = (double *)malloc(B->size * sizeof(double));

    struct timeval start = tic();

    for (int i = 0; i < 1; i++)
        iterativeRefinementGeneral(dense, B, X);

    printf("Iterative refinement time is %f\n", toc(start));

    sparseToDense(sparse, dense);    // overwrite factorized matrix to get original values for evaluation
    checkSolutionDense(dense, B, X, 0); // calculate |Ax-b|
}
