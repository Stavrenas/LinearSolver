#include "hip/hip_runtime.h"
void help()
{
    // Suppose that A is m x m sparse matrix represented by CSR format,
    // Assumption:
    // - handle is already created by hipsparseCreate(),
    // - (d_csrRowPtr, d_csrColInd, d_csrVal) is CSR of A on device memory,
    // - d_x is right hand side vector on device memory,
    // - d_y is solution vector on device memory.
    // - d_z is intermediate result on device memory.

    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    hipsparseMatDescr_t descr_U = 0;
    csrilu02Info_t info_M = 0;
    csrsv2Info_t info_L = 0;
    csrsv2Info_t info_U = 0;
    int pBufferSize_M;
    int pBufferSize_L;
    int pBufferSize_U;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const double alpha = 1.;
    const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_U = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_U = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has unit diagonal
    // - matrix U is base-1
    // - matrix U is upper triangular
    // - matrix U has non-unit diagonal
    hipsparseCreateMatDescr(&descr_M);
    hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

    hipsparseCreateMatDescr(&descr_L);
    hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
    hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

    hipsparseCreateMatDescr(&descr_U);
    hipsparseSetMatIndexBase(descr_U, HIPSPARSE_INDEX_BASE_ONE);
    hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
    hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for csrilu02 and two info's for csrsv2
    hipsparseCreateCsrilu02Info(&info_M);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    // step 3: query how much memory used in csrilu02 and csrsv2, and allocate the buffer
    hipsparseDcsrilu02_bufferSize(handle, m, nnz,
                                 descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, &pBufferSize_M);
    hipsparseDcsrsv2_bufferSize(handle, trans_L, m, nnz,
                               descr_L, d_csrVal, d_csrRowPtr, d_csrColInd, info_L, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(handle, trans_U, m, nnz,
                               descr_U, d_csrVal, d_csrRowPtr, d_csrColInd, info_U, &pBufferSize_U);

    pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void **)&pBuffer, pBufferSize);

    // step 4: perform analysis of incomplete Cholesky on M
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on U
    // The lower(upper) triangular part of M has the same sparsity pattern as L(U),
    // we can do analysis of csrilu0 and csrsv2 simultaneously.

    hipsparseDcsrilu02_analysis(handle, m, nnz, descr_M,
                               d_csrVal, d_csrRowPtr, d_csrColInd, info_M,
                               policy_M, pBuffer);
    status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
    {
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    hipsparseDcsrsv2_analysis(handle, trans_L, m, nnz, descr_L,
                             d_csrVal, d_csrRowPtr, d_csrColInd,
                             info_L, policy_L, pBuffer);

    hipsparseDcsrsv2_analysis(handle, trans_U, m, nnz, descr_U,
                             d_csrVal, d_csrRowPtr, d_csrColInd,
                             info_U, policy_U, pBuffer);

    // step 5: M = L * U
    hipsparseDcsrilu02(handle, m, nnz, descr_M,
                      d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer);
    status = hipsparseXcsrilu02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status)
    {
        printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }

    // step 6: solve L*z = x
    hipsparseDcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L,
                          d_csrVal, d_csrRowPtr, d_csrColInd, info_L,
                          d_x, d_z, policy_L, pBuffer);

    // step 7: solve U*y = z
    hipsparseDcsrsv2_solve(handle, trans_U, m, nnz, &alpha, descr_U,
                          d_csrVal, d_csrRowPtr, d_csrColInd, info_U,
                          d_z, d_y, policy_U, pBuffer);

    // step 6: free resources
    hipFree(pBuffer);
    hipsparseDestroyMatDescr(descr_M);
    hipsparseDestroyMatDescr(descr_L);
    hipsparseDestroyMatDescr(descr_U);
    hipsparseDestroyCsrilu02Info(info_M);
    hipsparseDestroyCsrsv2Info(info_L);
    hipsparseDestroyCsrsv2Info(info_U);
    hipsparseDestroy(handle);
}