#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "hipblas.h"
#include ""

extern "C"
{
#include "utilities.h"
#include "read.h"
#include "cudaUtilities.h"
}

void Cholesky(double **A, double *B, double *X, int size)
{

    double *Aserialized = (double *)malloc(size * size * sizeof(double));
    serializeMatrix(size, A, Aserialized);

    // INITIALIZE CUSOLVER
    hipsolverHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    hipsolverDnCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSetStream(cusolverHandle, stream);

    int info;
    int *d_info = nullptr; /* device error info */

    int Lwork = 0;               /* size of workspace */
    double *Workspace = nullptr; /* device workspace */

    /* step 2: COPY MATRICES TO DEVICE */
    double *Bcuda, *Acuda, *Xcuda;

    hipMalloc((void **)&Acuda, size * size * sizeof(double));
    hipMalloc((void **)&Bcuda, size * sizeof(double));
    hipMalloc((void **)&Xcuda, size * sizeof(double));

    hipMemcpy(Acuda, Aserialized, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Bcuda, B, size * sizeof(double), hipMemcpyHostToDevice);

    /* step 3: query working space */
    hipsolverDnDpotrf_bufferSize(cusolverHandle, uplo, size, Acuda, size, &Lwork);
    hipMalloc(&Workspace, sizeof(double) * Lwork);
    hipMalloc((void **)&d_info, sizeof(int));

    /* step 4: Cholesky factorization */
    hipsolverDnDpotrf(cusolverHandle, uplo, size, Acuda, size, Workspace, Lwork, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    if (0 > info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    else if (info > 0)
        exit(1);

    /* step 5: solve A*X = b */
    hipsolverDnDpotrs(cusolverHandle, uplo, size, 1, Acuda, size, Bcuda, size, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    if (0 > info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    hipMemcpy(X, Bcuda, sizeof(double) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* free resources */
    hipFree(Acuda);
    hipFree(Bcuda);
    hipFree(Xcuda);
    hipFree(d_info);
    hipFree(Workspace);

    // hipblasDestroy(handle);
    hipsolverDnDestroy(cusolverHandle);
    hipDeviceReset();

    free(A);
    free(B);
    free(X);
    free(Aserialized);
}

void iterativeRefinement(double **A, double *B, double *X, int size)
{

    double *Aserialized = (double *)malloc(size * size * sizeof(double));
    serializeMatrix(size, A, Aserialized);

    // INITIALIZE CUSOLVER
    hipsolverHandle_t cusolverHandle;
    hipStream_t stream = NULL;
    hipsolverStatus_t error;

    hipsolverDnCreate(&cusolverHandle);
    hipStreamCreate(&stream);
    hipsolverSetStream(cusolverHandle, stream);

    int info, *dipiv;
    int *d_info = nullptr; /* device error info */
    int *iters = (int *)malloc(sizeof(int));

    size_t Lwork = 0;          /* size of workspace */
    void *Workspace = nullptr; /* device workspace */

    /* step 2: COPY MATRICES TO DEVICE */
    double *Bcuda, *Acuda, *Xcuda;

    hipMalloc((void **)&Acuda, size * size * sizeof(double));
    hipMalloc((void **)&Bcuda, size * sizeof(double));
    hipMalloc((void **)&Xcuda, size * sizeof(double));

    hipMalloc((void **)&d_info, sizeof(int));
    hipMalloc((void **)&dipiv, size * size * sizeof(int));

    hipMemcpy(Acuda, Aserialized, size * size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Bcuda, B, size * sizeof(double), hipMemcpyHostToDevice);

    /* step 3: query working space */
    error = cusolverDnDHgesv_bufferSize(cusolverHandle, size, 1, Acuda, size, dipiv, Bcuda, size, Xcuda, size, Workspace, &Lwork);
    hipMalloc(&Workspace, Lwork);
    // printf("Error: %s\n", cudaGetErrorEnum(error));

    /* step 4: Iterative Refinement solution */
    error = cusolverDnDHgesv(cusolverHandle, size, 1, Acuda, size, dipiv, Bcuda, size, Xcuda, size, Workspace, Lwork, iters, d_info);
    hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    // printf("Error: %s\n", cudaGetErrorEnum(error));

    if (0 != info)
    {
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    // printf("Size of Workspace: %d\n", Lwork);
    // printf("Iterations: %d\n",*iters);
    hipMemcpy(X, Xcuda, sizeof(double) * size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    /* free resources */
    hipFree(Acuda);
    hipFree(Bcuda);
    hipFree(Xcuda);
    hipFree(d_info);
    hipFree(Workspace);

    // hipblasDestroy(handle);
    hipsolverDnDestroy(cusolverHandle);
    hipDeviceReset();

    free(Aserialized);
}


int main()
{
    int size = 273;
    double **A, *B, *Xcalculated, *X;

    // ALLOCATE MEMORY
    A = (double **)malloc(size * sizeof(double *));
    B = (double *)malloc(size * sizeof(double));
    Xcalculated = (double *)malloc(size * sizeof(double));
    X = (double *)malloc(size * sizeof(double));

    for (int i = 0; i < size; i++)
        A[i] = (double *)malloc(size * sizeof(double));

    // READ MATRICES
    readSquareMatrix("A.txt", size, A);
    readVector("B.txt", size, B);
    readVector("X.txt", size, X);

    struct timeval start = tic();

    iterativeRefinement(A, B, Xcalculated, size);

    printf("Iterative refinement time is %f\n", toc(start));

    double thres = 1e-10;
    if (compareVectors(size, X, Xcalculated, thres))
        printf("Solution is True\n");
    else
        printf("Solution is False\n");

    // start = tic();

    // printf("\n\n");

    // Cholesky(A,B,Xcalculated,size);

    // printf("Cholesky factorization time is %f\n",toc(start));

    // if (compareVectors(size, X, Xcalculated,thres))
    //     printf("Solution is True\n");
    // else
    //     printf("Solution is False\n");
}

